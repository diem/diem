#include "hip/hip_runtime.h"
// Cuckoo Cycle, a memory-hard proof-of-work
// Copyright (c) 2013-2016 John Tromp

// The edge-trimming memory optimization is due to Dave Andersen
// http://da-data.blogspot.com/2014/03/a-public-review-of-cuckoo-cycle.html

#include <stdint.h>
#include <string.h>
#include "cuckatoo.h"
#include "../crypto/siphash.cuh"
#include "graph.hpp"

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <set>

#ifndef MAXSOLS
#define MAXSOLS 4
#endif

typedef uint8_t u8;
typedef uint16_t u16;
typedef uint64_t u64; // save some typing

// algorithm parameters
#ifndef PART_BITS
// #bits used to partition edge set processing to save memory
// a value of 0 does no partitioning and is fastest
// a value of 1 partitions in two at about 33% slowdown
// higher values are not that interesting
#define PART_BITS 0
#endif

#ifndef IDXSHIFT
#define IDXSHIFT (PART_BITS + 8)
#endif
#define MAXEDGES (NEDGES >> IDXSHIFT)

const u64 edgeBytes = NEDGES/8;
const u64 nodeBytes = (NEDGES>>PART_BITS)/8;
const u32 PART_MASK = (1 << PART_BITS) - 1;
const u32 NONPART_BITS = EDGEBITS - PART_BITS;
const word_t NONPART_MASK = ((word_t)1 << NONPART_BITS) - 1;

#define NODEBITS (EDGEBITS + 1)
#define NNODES2 (2 * NEDGES)
#define NODE2MASK (NNODES2-1)

#define checkCudaErrors_V(ans) ({if (gpuAssert((ans), __FILE__, __LINE__) != hipSuccess) return;})
#define checkCudaErrors_N(ans) ({if (gpuAssert((ans), __FILE__, __LINE__) != hipSuccess) return NULL;})
#define checkCudaErrors(ans) ({int retval = gpuAssert((ans), __FILE__, __LINE__); if (retval != hipSuccess) return retval;})

inline int gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
  int device_id;
  hipGetDevice(&device_id);
  if (code != hipSuccess) {
    snprintf(LAST_ERROR_REASON, MAX_NAME_LEN, "Device %d GPUassert: %s %s %d", device_id, hipGetErrorString(code), file, line);
    hipDeviceReset();
    if (abort) return code;
  }
  return code;
}

// set that starts out full and gets reset by threads on disjoint words
class shrinkingset {
public:
  u32 *bits;
  __device__ void reset(word_t n) {
    bits[n/32] |= 1 << (n%32);
  }
  __device__ bool test(word_t n) const {
    return !((bits[n/32] >> (n%32)) & 1);
  }
  __device__ u32 block(word_t n) const {
    return ~bits[n/32];
  }
};

class biitmap {
public:
  u32 *bits;
  __device__ void set(word_t n) {
    atomicOr(&bits[n/32], 1 << (n%32));
  }
  __device__ bool test(word_t n) const {
    return (bits[n/32] >> (n%32)) & 1;
  }
};

struct trimparams {
  u16 ntrims;
  u16 blocks;
  u16 tpb;

  trimparams() {
    ntrims      = 128;
    blocks      = 128;
    tpb         = 128;
  }
};

__global__ void count_node_deg(siphash_keys &sipkeys, shrinkingset &alive, biitmap &nonleaf, u32 uorv, u32 part) {
  const int nthreads = blockDim.x * gridDim.x;
  const int id = blockIdx.x * blockDim.x + threadIdx.x;
  for (word_t block = id*32; block < NEDGES; block += nthreads*32) {
    u32 alive32 = alive.block(block);
    for (word_t nonce = block-1; alive32; ) { // -1 compensates for 1-based ffs
      u32 ffs = __ffs(alive32);
      nonce += ffs; alive32 >>= ffs;
      word_t u = dipnode(sipkeys, nonce, uorv);
      if ((u >> NONPART_BITS) == part) {
        nonleaf.set(u & NONPART_MASK);
      }
    }
  }
}

__global__ void kill_leaf_edges(siphash_keys &sipkeys, shrinkingset &alive, biitmap &nonleaf, u32 uorv, u32 part) {
  const int nthreads = blockDim.x * gridDim.x;
  const int id = blockIdx.x * blockDim.x + threadIdx.x;
  for (word_t block = id*32; block < NEDGES; block += nthreads*32) {
    u32 alive32 = alive.block(block);
    for (word_t nonce = block-1; alive32; ) { // -1 compensates for 1-based ffs
      u32 ffs = __ffs(alive32);
      nonce += ffs; alive32 >>= ffs;
      word_t u = dipnode(sipkeys, nonce, uorv);
      if ((u >> NONPART_BITS) == part && !nonleaf.test((u & NONPART_MASK) ^ 1)) {
        alive.reset(nonce);
      }
    }
  }
}

// maintains set of trimmable edges
struct edgetrimmer {
  trimparams tp;
  edgetrimmer *dt;
  shrinkingset alive;
  biitmap nonleaf;
  siphash_keys sipkeys, *dipkeys;
  bool abort;
  bool initsuccess = false;

  edgetrimmer(const trimparams _tp) : tp(_tp) {
    checkCudaErrors_V(hipMalloc((void**)&dt, sizeof(edgetrimmer)));
    checkCudaErrors_V(hipMalloc((void**)&dipkeys, sizeof(siphash_keys)));
    checkCudaErrors_V(hipMalloc((void**)&alive.bits, edgeBytes));
    checkCudaErrors_V(hipMalloc((void**)&nonleaf.bits, nodeBytes));
    hipMemcpy(dt, this, sizeof(edgetrimmer), hipMemcpyHostToDevice);
    initsuccess = true;
  }
  ~edgetrimmer() {
    checkCudaErrors_V(hipFree(nonleaf.bits));
    checkCudaErrors_V(hipFree(alive.bits));
    checkCudaErrors_V(hipFree(dipkeys));
    checkCudaErrors_V(hipFree(dt));
    hipDeviceReset();
  }
  bool trim() {
    hipMemcpy(dipkeys, &sipkeys, sizeof(sipkeys), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    checkCudaErrors(hipMemset(alive.bits, 0, edgeBytes));
    for (u32 round=0; round < tp.ntrims; round++) {
      for (u32 part = 0; part <= PART_MASK; part++) {
        checkCudaErrors(hipMemset(nonleaf.bits, 0, nodeBytes));
        if (abort) return false;
        count_node_deg<<<tp.blocks,tp.tpb>>>(*dipkeys, dt->alive, dt->nonleaf, round&1, part);
        if (abort) return false;
        kill_leaf_edges<<<tp.blocks,tp.tpb>>>(*dipkeys, dt->alive, dt->nonleaf, round&1, part);
        if (abort) return false;
      }
    }
    return true;
  }
};

struct solver_ctx {
public:
  edgetrimmer trimmer;
  bool mutatenonce;
  graph<word_t> cg;
  u64 *bits;
  proof sols[MAXSOLS];

  solver_ctx(const trimparams tp, bool mutate_nonce) : trimmer(tp), cg(MAXEDGES, MAXEDGES, MAXSOLS, IDXSHIFT) {
    bits = new u64[NEDGES/64];
    mutatenonce = mutate_nonce;
  }

  void setheadernonce(char * const headernonce, const u32 len, const u32 nonce) {
    if (mutatenonce) {
      ((u32 *)headernonce)[len/sizeof(u32)-1] = htole32(nonce); // place nonce at end
    }
    setheader(headernonce, len, &trimmer.sipkeys);
  }
  ~solver_ctx() {
    delete[] bits;
  }

  void findcycles() {
    cg.reset();
    for (word_t block = 0; block < NEDGES; block += 64) {
      u64 alive64 = ~bits[block/64];
      for (word_t nonce = block-1; alive64; ) { // -1 compensates for 1-based ffs
        u32 ffs = __builtin_ffsll(alive64);
        nonce += ffs; alive64 >>= ffs;
        word_t u=sipnode(&trimmer.sipkeys, nonce, 0), v=sipnode(&trimmer.sipkeys, nonce, 1);
	cg.add_compress_edge(u, v);
        if (ffs & 64) break; // can't shift by 64
      }
    }
  }

  int solve() {
    u64 time0, time1;
    u32 timems,timems2;
    time0 = timestamp();

    trimmer.abort = false;
    if (!trimmer.trim()) // trimmer aborted
      return 0;

    hipMemcpy(bits, trimmer.alive.bits, edgeBytes, hipMemcpyDeviceToHost);
    u32 nedges = 0;
    for (int i = 0; i < NEDGES/64; i++)
      nedges += __builtin_popcountll(~bits[i]);
    if (nedges >= MAXEDGES) {
      print_log("overloaded! exiting...");
      exit(0);
    }
    time1 = timestamp(); timems  = (time1 - time0) / 1000000;
    time0 = timestamp();
    findcycles();
    time1 = timestamp(); timems2 = (time1 - time0) / 1000000;

    print_log("%d trims %d ms %d edges %d ms total %d ms\n", trimmer.tp.ntrims, timems, nedges, timems2, timems+timems2);

    for (u32 s=0; s < cg.nsols; s++) {
      u32 j = 0, nalive = 0;
      for (word_t block = 0; block < NEDGES; block += 64) {
        u64 alive64 = ~bits[block/64];
        for (word_t nonce = block-1; alive64; ) { // -1 compensates for 1-based ffs
          u32 ffs = __builtin_ffsll(alive64);
          nonce += ffs; alive64 >>= ffs;
          if (nalive++ == cg.sols[s][j]) {
            sols[s][j] = nonce;
            if (++j == PROOFSIZE)
              goto uncompressed;
          }
          if (ffs & 64) break; // can't shift by 64
        }
      }
      uncompressed: ;
    }
    return cg.nsols;
  }

  void abort() {
    trimmer.abort = true;
  }
};

// arbitrary length of header hashed into siphash key
#define HEADERLEN 80

typedef std::pair<word_t,word_t> edge;

#include <unistd.h>

typedef solver_ctx SolverCtx;

CALL_CONVENTION int run_solver(SolverCtx* ctx,
                               char* header,
                               int header_length,
                               u32 nonce,
                               u32 range,
                               SolverSolutions *solutions,
                               SolverStats *stats
                               )
{
  u64 time0, time1;
  u32 timems;
  u32 sumnsols = 0;
  int device_id;
  if (stats != NULL) {
    hipGetDevice(&device_id);
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, stats->device_id);
    stats->device_id = device_id;
    stats->edge_bits = EDGEBITS;
    strncpy(stats->device_name, props.name, MAX_NAME_LEN);
  }

  if (ctx == NULL || !ctx->trimmer.initsuccess){
    print_log("Error initialising trimmer. Aborting.\n");
    print_log("Reason: %s\n", LAST_ERROR_REASON);
    if (stats != NULL) {
       stats->has_errored = true;
       strncpy(stats->error_reason, LAST_ERROR_REASON, MAX_NAME_LEN);
    }
    return 0;
  }

  for (u32 r = 0; r < range; r++) {
    time0 = timestamp();
    ctx->setheadernonce(header, header_length, nonce + r);
    print_log("nonce %d k0 k1 k2 k3 %llx %llx %llx %llx\n", nonce+r, ctx->trimmer.sipkeys.k0, ctx->trimmer.sipkeys.k1, ctx->trimmer.sipkeys.k2, ctx->trimmer.sipkeys.k3);
    u32 nsols = ctx->solve();
    time1 = timestamp(); timems = (time1 - time0) / 1000000;
    print_log("Time: %d ms\n", timems);
    for (u32 s = 0; s < nsols; s++) {
      print_log("Solution");
      for (u32 j = 0; j < PROOFSIZE; j++)
        print_log(" %x", ctx->sols[s][j]);
      print_log("\n");
      if (solutions != NULL){
        solutions->edge_bits = EDGEBITS;
        solutions->num_sols++;
        solutions->sols[sumnsols+s].nonce = nonce + r;
        for (u32 i = 0; i < PROOFSIZE; i++)
          solutions->sols[sumnsols+s].proof[i] = (u64) ctx->sols[s][i];
      }
      int pow_rc = verify(ctx->sols[s], &ctx->trimmer.sipkeys);
      if (pow_rc == POW_OK) {
        print_log("Verified with cyclehash ");
        unsigned char cyclehash[32];
        blake2b((void *)cyclehash, sizeof(cyclehash), (const void *)ctx->cg.sols[s], sizeof(ctx->cg.sols[0]), 0, 0);
        for (int i=0; i<32; i++)
          print_log("%02x", cyclehash[i]);
        print_log("\n");
      } else {
        print_log("FAILED due to %s\n", errstr[pow_rc]);
      }
    }
    sumnsols += nsols;
    if (stats != NULL) {
        stats->last_start_time = time0;
        stats->last_end_time = time1;
        stats->last_solution_time = time1 - time0;
    }
  }
  print_log("%d total solutions\n", sumnsols);
	return 0;
}

CALL_CONVENTION SolverCtx* create_solver_ctx(SolverParams* params) {
  trimparams tp;
  tp.ntrims = params->ntrims;
	tp.blocks = params->blocks;
	tp.tpb = params->tpb;

  hipDeviceProp_t prop;
  checkCudaErrors_N(hipGetDeviceProperties(&prop, params->device));

  assert(tp.tpb <= prop.maxThreadsPerBlock);

  hipSetDevice(params->device);

  SolverCtx* ctx = new SolverCtx(tp, params->mutate_nonce);

  return ctx;
}

CALL_CONVENTION void destroy_solver_ctx(SolverCtx* ctx) {
  delete ctx;
}

CALL_CONVENTION void stop_solver(SolverCtx* ctx) {
  ctx->abort();
}

CALL_CONVENTION void fill_default_params(SolverParams* params) {
  trimparams tp;
  params->device = 0;
  params->ntrims = tp.ntrims;
  params->blocks = tp.blocks;
  params->tpb = tp.tpb;
}

int main(int argc, char **argv) {
  trimparams tp;
  u32 nonce = 0;
  u32 range = 1;
  u32 device = 0;
  char header[HEADERLEN];
  u32 len;
  int c;

  // set defaults
  SolverParams params;
  fill_default_params(&params);

  memset(header, 0, sizeof(header));
  while ((c = getopt (argc, argv, "sb:d:h:n:m:r:t:")) != -1) {
    switch (c) {
      case 's':
        print_log("SYNOPSIS\n  lcuda%d [-d device] [-h hexheader] [-m trims] [-n nonce] [-r range] [-b blocks] [-t threads]\n", NODEBITS);
        print_log("DEFAULTS\n  cuda%d -d %d -h \"\" -m %d -n %d -r %d -b %d -t %d\n", NODEBITS, device, tp.ntrims, nonce, range, tp.blocks, tp.tpb);
        exit(0);
      case 'd':
        device = params.device = atoi(optarg);
        break;
      case 'h':
        len = strlen(optarg)/2;
        assert(len <= sizeof(header));
        for (u32 i=0; i<len; i++)
          sscanf(optarg+2*i, "%2hhx", header+i); // hh specifies storage of a single byte
        break;
      case 'n':
        nonce = atoi(optarg);
        break;
      case 'm':
        params.ntrims = atoi(optarg);
        break;
      case 'r':
        range = atoi(optarg);
        break;
      case 'b':
        params.blocks = atoi(optarg);
        break;
      case 't':
        params.tpb = atoi(optarg);
        break;
    }
  }
  int nDevices;
  checkCudaErrors(hipGetDeviceCount(&nDevices));
  assert(device < nDevices);
  hipDeviceProp_t prop;
  checkCudaErrors(hipGetDeviceProperties(&prop, device));
  u64 dbytes = prop.totalGlobalMem;
  int dunit;
  for (dunit=0; dbytes >= 10240; dbytes>>=10,dunit++) ;
  print_log("%s with %d%cB @ %d bits x %dMHz\n", prop.name, (u32)dbytes, " KMGT"[dunit], prop.memoryBusWidth, prop.memoryClockRate/1000);

  print_log("Looking for %d-cycle on cuckatoo%d(\"%s\",%d", PROOFSIZE, EDGEBITS, header, nonce);
  if (range > 1)
    print_log("-%d", nonce+range-1);
  print_log(") with 50%% edges, %d trims, %d threads %d per block\n", tp.ntrims, tp.blocks*tp.tpb, tp.tpb);

  int edgeUnit=0, nodeUnit=0;
  u64 eb = edgeBytes, nb = nodeBytes;
  for (; eb >= 1024; eb>>=10) edgeUnit++;
  for (; nb >= 1024; nb>>=10) nodeUnit++;
  print_log("Using %d%cB edge and %d%cB node memory.\n", (int)eb, " KMGT"[edgeUnit], (int)nb, " KMGT"[nodeUnit]);

  SolverCtx* ctx = create_solver_ctx(&params);

  run_solver(ctx, header, sizeof(header), nonce, range, NULL, NULL);

  return 0;
}
