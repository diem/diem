#include "hip/hip_runtime.h"
// Cuckoo Cycle, a memory-hard proof-of-work by John Tromp
// Copyright (c) 2018 Jiri Vadura (photon) and John Tromp
// This software is covered by the FAIR MINING license

#include <stdio.h>
#include <string.h>
#include <vector>
#include <assert.h>
#include "cuckoo.h"
#include "../crypto/siphash.cuh"
#include "../crypto/blake2.h"

typedef uint8_t u8;
typedef uint16_t u16;

typedef u32 node_t;
typedef u64 nonce_t;

#ifndef XBITS
#define XBITS 6
#endif

#define NODEBITS (EDGEBITS + 1)
#define NNODES ((node_t)1 << NODEBITS)
#define NODEMASK (NNODES - 1)

const u32 NX        = 1 << XBITS;
const u32 NX2       = NX * NX;
const u32 XMASK     = NX - 1;
const u32 YBITS     = XBITS;
const u32 NY        = 1 << YBITS;
const u32 YZBITS    = EDGEBITS - XBITS;
const u32 ZBITS     = YZBITS - YBITS;
const u32 NZ        = 1 << ZBITS;
const u32 ZMASK     = NZ - 1;

#ifndef NEPS_A
#define NEPS_A 133
#endif
#ifndef NEPS_B
#define NEPS_B 88
#endif
#define NEPS 128

const u32 EDGES_A = NZ * NEPS_A / NEPS;
const u32 EDGES_B = NZ * NEPS_B / NEPS;

const u32 ROW_EDGES_A = EDGES_A * NY;
const u32 ROW_EDGES_B = EDGES_B * NY;

// Number of Parts of BufferB, all but one of which will overlap BufferA
#ifndef NB
#define NB 2
#endif

#ifndef NA
#define NA  ((NB * NEPS_A + NEPS_B-1) / NEPS_B)
#endif

__constant__ uint2 recoveredges[PROOFSIZE];
__constant__ uint2 e0 = {0,0};

__device__ uint2 make_Edge(const u32 nonce, const uint2 dummy, const u32 node0, const u32 node1) {
   return make_uint2(node0, node1);
}

__device__ uint2 make_Edge(const uint2 edge, const uint2 dummy, const u32 node0, const u32 node1) {
   return edge;
}

__device__ u32 make_Edge(const u32 nonce, const u32 dummy, const u32 node0, const u32 node1) {
   return nonce;
}

#ifndef FLUSHA // should perhaps be in trimparams and passed as template parameter
#define FLUSHA 16
#endif

template<int maxOut, typename EdgeOut>
__global__ void SeedA(const siphash_keys &sipkeys, ulonglong4 * __restrict__ buffer, u32 * __restrict__ indexes) {
  const int group = blockIdx.x;
  const int dim = blockDim.x;
  const int lid = threadIdx.x;
  const int gid = group * dim + lid;
  const int nthreads = gridDim.x * dim;
  const int FLUSHA2 = 2*FLUSHA;

  __shared__ EdgeOut tmp[NX][FLUSHA2]; // needs to be ulonglong4 aligned
  const int TMPPERLL4 = sizeof(ulonglong4) / sizeof(EdgeOut);
  __shared__ int counters[NX];

  for (int row = lid; row < NX; row += dim)
    counters[row] = 0;
  __syncthreads();

  const int col = group % NX;
  const int loops = NEDGES / nthreads;
  for (int i = 0; i < loops; i++) {
    u32 nonce = gid * loops + i;
    u32 node1, node0 = dipnode(sipkeys, (u64)nonce, 0);
    if (sizeof(EdgeOut) == sizeof(uint2))
      node1 = dipnode(sipkeys, (u64)nonce, 1);
    int row = node0 >> YZBITS;
    int counter = min((int)atomicAdd(counters + row, 1), (int)(FLUSHA2-1));
    tmp[row][counter] = make_Edge(nonce, tmp[0][0], node0, node1);
    __syncthreads();
    if (counter == FLUSHA-1) {
      int localIdx = min(FLUSHA2, counters[row]);
      int newCount = localIdx % FLUSHA;
      int nflush = localIdx - newCount;
      u32 grp = row * NX + col;
      int cnt = min((int)atomicAdd(indexes + grp, nflush), (int)(maxOut - nflush));
      for (int i = 0; i < nflush; i += TMPPERLL4)
        buffer[((u64)grp * maxOut + cnt + i) / TMPPERLL4] = *(ulonglong4 *)(&tmp[row][i]);
      for (int t = 0; t < newCount; t++) {
        tmp[row][t] = tmp[row][t + nflush];
      }
      counters[row] = newCount;
    }
    __syncthreads();
  }
  EdgeOut zero = make_Edge(0, tmp[0][0], 0, 0);
  for (int row = lid; row < NX; row += dim) {
    int localIdx = min(FLUSHA2, counters[row]);
    u32 grp = row * NX + col;
    for (int j = localIdx; j % TMPPERLL4; j++)
      tmp[row][j] = zero;
    for (int i = 0; i < localIdx; i += TMPPERLL4) {
      int cnt = min((int)atomicAdd(indexes + grp, TMPPERLL4), (int)(maxOut - TMPPERLL4));
      buffer[((u64)grp * maxOut + cnt) / TMPPERLL4] = *(ulonglong4 *)(&tmp[row][i]);
    }
  }
}

template <typename Edge> __device__ bool null(Edge e);

__device__ bool null(u32 nonce) {
  return nonce == 0;
}

__device__ bool null(uint2 nodes) {
  return nodes.x == 0 && nodes.y == 0;
}

#ifndef FLUSHB
#define FLUSHB 8
#endif

template<int maxOut, typename EdgeOut>
__global__ void SeedB(const siphash_keys &sipkeys, const EdgeOut * __restrict__ source, ulonglong4 * __restrict__ dst, const u32 * __restrict__ srcIdx, u32 * __restrict__ dstIdx) {
  const int group = blockIdx.x;
  const int dim = blockDim.x;
  const int lid = threadIdx.x;
  const int FLUSHB2 = 2 * FLUSHB;

  __shared__ EdgeOut tmp[NX][FLUSHB2];
  const int TMPPERLL4 = sizeof(ulonglong4) / sizeof(EdgeOut);
  __shared__ int counters[NX];

  // if (group>=0&&lid==0) print_log("group  %d  -\n", group);
  for (int col = lid; col < NX; col += dim)
    counters[col] = 0;
  __syncthreads();
  const int row = group / NX;
  const int bucketEdges = min((int)srcIdx[group], (int)maxOut);
  const int loops = (bucketEdges + dim-1) / dim;
  for (int loop = 0; loop < loops; loop++) {
    int col;
    int counter = 0;
    const int edgeIndex = loop * dim + lid;
    if (edgeIndex < bucketEdges) {
      const int index = group * maxOut + edgeIndex;
      EdgeOut edge = __ldg(&source[index]);
      if (!null(edge)) {
        u32 node1 = endpoint(sipkeys, edge, 0);
        col = (node1 >> ZBITS) & XMASK;
        counter = min((int)atomicAdd(counters + col, 1), (int)(FLUSHB2-1));
        tmp[col][counter] = edge;
      }
    }
    __syncthreads();
    if (counter == FLUSHB-1) {
      int localIdx = min(FLUSHB2, counters[col]);
      int newCount = localIdx % FLUSHB;
      int nflush = localIdx - newCount;
      u32 grp = row * NX + col;
      int cnt = min((int)atomicAdd(dstIdx + grp, nflush), (int)(maxOut - nflush));
      for (int i = 0; i < nflush; i += TMPPERLL4)
        dst[((u64)grp * maxOut + cnt + i) / TMPPERLL4] = *(ulonglong4 *)(&tmp[col][i]);
      for (int t = 0; t < newCount; t++) {
        tmp[col][t] = tmp[col][t + nflush];
      }
      counters[col] = newCount;
    }
    __syncthreads(); 
  }
  EdgeOut zero = make_Edge(0, tmp[0][0], 0, 0);
  for (int col = lid; col < NX; col += dim) {
    int localIdx = min(FLUSHB2, counters[col]);
    u32 grp = row * NX + col;
    for (int j = localIdx; j % TMPPERLL4; j++)
      tmp[col][j] = zero;
    for (int i = 0; i < localIdx; i += TMPPERLL4) {
      int cnt = min((int)atomicAdd(dstIdx + grp, TMPPERLL4), (int)(maxOut - TMPPERLL4));
      dst[((u64)grp * maxOut + cnt) / TMPPERLL4] = *(ulonglong4 *)(&tmp[col][i]);
    }
  }
}

__device__ __forceinline__  void Increase2bCounter(u32 *ecounters, const int bucket) {
  int word = bucket >> 5;
  unsigned char bit = bucket & 0x1F;
  u32 mask = 1 << bit;

  u32 old = atomicOr(ecounters + word, mask) & mask;
  if (old)
    atomicOr(ecounters + word + NZ/32, mask);
}

__device__ __forceinline__  bool Read2bCounter(u32 *ecounters, const int bucket) {
  int word = bucket >> 5;
  unsigned char bit = bucket & 0x1F;
  u32 mask = 1 << bit;

  return (ecounters[word + NZ/32] & mask) != 0;
}

template <typename Edge> u32 __device__ endpoint(const siphash_keys &sipkeys, Edge e, int uorv);

__device__ u32 endpoint(const siphash_keys &sipkeys, u32 nonce, int uorv) {
  return dipnode(sipkeys, nonce, uorv);
}

__device__ u32 endpoint(const siphash_keys &sipkeys, uint2 nodes, int uorv) {
  return uorv ? nodes.y : nodes.x;
}

template<int NP, int maxIn, typename EdgeIn, int maxOut, typename EdgeOut>
__global__ void Round(const int round, const siphash_keys &sipkeys, const EdgeIn * __restrict__ src, EdgeOut * __restrict__ dst, const u32 * __restrict__ srcIdx, u32 * __restrict__ dstIdx) {
  const int group = blockIdx.x;
  const int dim = blockDim.x;
  const int lid = threadIdx.x;
  const int COUNTERWORDS = NZ / 16; // 16 2-bit counters per 32-bit word

  __shared__ u32 ecounters[COUNTERWORDS];

  for (int i = lid; i < COUNTERWORDS; i += dim)
    ecounters[i] = 0;
  __syncthreads();

  for (int i = 0; i < NP; i++, src += NX2 * maxIn, srcIdx += NX2) {
    const int edgesInBucket = min(srcIdx[group], maxIn);
    // if (!group && !lid) printf("round %d size  %d\n", round, edgesInBucket);
    const int loops = (edgesInBucket + dim-1) / dim;
    for (int loop = 0; loop < loops; loop++) {
      const int lindex = loop * dim + lid;
      if (lindex < edgesInBucket) {
        const int index = maxIn * group + lindex;
        EdgeIn edge = __ldg(&src[index]);
        if (null(edge)) continue;
        u32 node = endpoint(sipkeys, edge, round&1);
        Increase2bCounter(ecounters, node & ZMASK);
      }
    }
  }
  __syncthreads();
  src -= NP * NX2 * maxIn; srcIdx -= NP * NX2;
  for (int i = 0; i < NP; i++, src += NX2 * maxIn, srcIdx += NX2) {
    const int edgesInBucket = min(srcIdx[group], maxIn);
    const int loops = (edgesInBucket + dim-1) / dim;
    for (int loop = 0; loop < loops; loop++) {
      const int lindex = loop * dim + lid;
      if (lindex < edgesInBucket) {
        const int index = maxIn * group + lindex;
        EdgeIn edge = __ldg(&src[index]);
        if (null(edge)) continue;
        u32 node0 = endpoint(sipkeys, edge, round&1);
        if (Read2bCounter(ecounters, node0 & ZMASK)) {
          u32 node1 = endpoint(sipkeys, edge, (round&1)^1);
          const int bucket = node1 >> ZBITS;
          const int bktIdx = min(atomicAdd(dstIdx + bucket, 1), maxOut - 1);
          dst[bucket * maxOut + bktIdx] = (round&1) ? make_Edge(edge, *dst, node1, node0) : make_Edge(edge, *dst, node0, node1);
        }
      }
    }
  }
  // if (group==0&&lid==0) print_log("round %d cnt(0,0) %d\n", round, srcIdx[0]);
}

template<int maxIn>
__global__ void Tail(const uint2 *source, uint2 *destination, const u32 *srcIdx, u32 *dstIdx) {
  const int lid = threadIdx.x;
  const int group = blockIdx.x;
  const int dim = blockDim.x;
  int myEdges = srcIdx[group];
  __shared__ int destIdx;

  if (lid == 0)
    destIdx = atomicAdd(dstIdx, myEdges);
  __syncthreads();
  for (int i = lid; i < myEdges; i += dim)
    destination[destIdx + i] = source[group * maxIn + i];
}

#define checkCudaErrors_V(ans) ({if (gpuAssert((ans), __FILE__, __LINE__) != hipSuccess) return;})
#define checkCudaErrors_N(ans) ({if (gpuAssert((ans), __FILE__, __LINE__) != hipSuccess) return NULL;})
#define checkCudaErrors(ans) ({int retval = gpuAssert((ans), __FILE__, __LINE__); if (retval != hipSuccess) return retval;})

inline int gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
  int device_id;
  hipGetDevice(&device_id);
  if (code != hipSuccess) {
    snprintf(LAST_ERROR_REASON, MAX_NAME_LEN, "Device %d GPUassert: %s %s %d", device_id, hipGetErrorString(code), file, line);
    hipDeviceReset();
    if (abort) return code;
  }
  return code;
}

__global__ void Recovery(const siphash_keys &sipkeys, ulonglong4 *buffer, int *indexes) {
  const int gid = blockDim.x * blockIdx.x + threadIdx.x;
  const int lid = threadIdx.x;
  const int nthreads = blockDim.x * gridDim.x;
  const int loops = NEDGES / nthreads;
  __shared__ u32 nonces[PROOFSIZE];
  
  if (lid < PROOFSIZE) nonces[lid] = 0;
  __syncthreads();
  for (int i = 0; i < loops; i++) {
    u64 nonce = gid * loops + i;
    u64 u = dipnode(sipkeys, nonce, 0);
    u64 v = dipnode(sipkeys, nonce, 1);
    for (int i = 0; i < PROOFSIZE; i++) {
      if (recoveredges[i].x == u && recoveredges[i].y == v)
        nonces[i] = nonce;
    }
  }
  __syncthreads();
  if (lid < PROOFSIZE) {
    if (nonces[lid] > 0)
      indexes[lid] = nonces[lid];
  }
}

struct blockstpb {
  u16 blocks;
  u16 tpb;
};

struct trimparams {
  u16 expand;
  u16 ntrims;
  blockstpb genA;
  blockstpb genB;
  blockstpb trim;
  blockstpb tail;
  blockstpb recover;

  trimparams() {
    expand              =    0;
    ntrims              =  176;
    genA.blocks         = 4096;
    genA.tpb            =  256;
    genB.blocks         =  NX2;
    genB.tpb            =  128;
    trim.blocks         =  NX2;
    trim.tpb            =  512;
    tail.blocks         =  NX2;
    tail.tpb            = 1024;
    recover.blocks      = 1024;
    recover.tpb         = 1024;
  }
};

typedef u32 proof[PROOFSIZE];

// maintains set of trimmable edges
struct edgetrimmer {
  trimparams tp;
  edgetrimmer *dt;
  size_t sizeA, sizeB;
  const size_t indexesSize = NX * NY * sizeof(u32);
  u8 *bufferA;
  u8 *bufferB;
  u8 *bufferAB;
  u32 *indexesE[1+NB];
  u32 nedges;
  u32 *uvnodes;
  siphash_keys sipkeys, *dipkeys;
  bool abort;
  bool initsuccess = false;

  edgetrimmer(const trimparams _tp) : tp(_tp) {
    checkCudaErrors_V(hipMalloc((void**)&dt, sizeof(edgetrimmer)));
    checkCudaErrors_V(hipMalloc((void**)&uvnodes, PROOFSIZE * 2 * sizeof(u32)));
    checkCudaErrors_V(hipMalloc((void**)&dipkeys, sizeof(siphash_keys)));
    for (int i = 0; i < 1+NB; i++) {
      checkCudaErrors_V(hipMalloc((void**)&indexesE[i], indexesSize));
    }
    sizeA = ROW_EDGES_A * NX * (tp.expand > 0 ? sizeof(u32) : sizeof(uint2));
    sizeB = ROW_EDGES_B * NX * (tp.expand > 1 ? sizeof(u32) : sizeof(uint2));
    const size_t bufferSize = sizeA + sizeB / NB;
    if (tp.expand != 1)
      assert(bufferSize >= sizeB + sizeB / NB / 2); // ensure enough space for Round 1
    checkCudaErrors_V(hipMalloc((void**)&bufferA, bufferSize));
    bufferAB = bufferA + sizeB / NB;
    bufferB  = bufferA + bufferSize - sizeB;
    assert(bufferA + sizeA == bufferB + sizeB * (NB-1) / NB); // ensure alignment of overlap
    hipMemcpy(dt, this, sizeof(edgetrimmer), hipMemcpyHostToDevice);
    initsuccess = true;
  }
  u64 globalbytes() const {
    return (sizeA+sizeB/NB) + (1+NB) * indexesSize + sizeof(siphash_keys) + PROOFSIZE * 2 * sizeof(u32) + sizeof(edgetrimmer);
  }
  ~edgetrimmer() {
    checkCudaErrors_V(hipFree(bufferA));
    for (int i = 0; i < 1+NB; i++) {
      checkCudaErrors_V(hipFree(indexesE[i]));
    }
    checkCudaErrors_V(hipFree(dipkeys));
    checkCudaErrors_V(hipFree(uvnodes));
    checkCudaErrors_V(hipFree(dt));
    hipDeviceReset();
  }
  u32 trim() {
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start)); checkCudaErrors(hipEventCreate(&stop));
  
    hipMemcpy(dipkeys, &sipkeys, sizeof(sipkeys), hipMemcpyHostToDevice);
  
    hipDeviceSynchronize();
    float durationA, durationB;
    hipEventRecord(start, NULL);
  
    hipMemset(indexesE[1], 0, indexesSize);

    if (tp.expand == 0) {
      SeedA<EDGES_A, uint2><<<tp.genA.blocks, tp.genA.tpb>>>(*dipkeys, (ulonglong4*)bufferAB, (u32 *)indexesE[1]);
    } else {
      SeedA<EDGES_A,   u32><<<tp.genA.blocks, tp.genA.tpb>>>(*dipkeys, (ulonglong4*)bufferAB, (u32 *)indexesE[1]);
    }
  
    checkCudaErrors(hipDeviceSynchronize()); hipEventRecord(stop, NULL);
    hipEventSynchronize(stop); hipEventElapsedTime(&durationA, start, stop);
    if (abort) return false;
    hipEventRecord(start, NULL);
  
    hipMemset(indexesE[0], 0, indexesSize);

    size_t qA = sizeA/NA;
    size_t qE = NX2 / NA;
    for (u32 i = 0; i < NA; i++) {
      if (tp.expand == 0) {
        SeedB<EDGES_A, uint2><<<tp.genB.blocks/NA, tp.genB.tpb>>>(*dipkeys, (const uint2 *)(bufferAB+i*qA), (ulonglong4*)(bufferA+i*qA), indexesE[1]+i*qE, indexesE[0]+i*qE);
      } else {
        SeedB<EDGES_A,   u32><<<tp.genB.blocks/NA, tp.genB.tpb>>>(*dipkeys, (const   u32 *)(bufferAB+i*qA), (ulonglong4*)(bufferA+i*qA), indexesE[1]+i*qE, indexesE[0]+i*qE);
      }
      if (abort) return false;
    }

    checkCudaErrors(hipDeviceSynchronize()); hipEventRecord(stop, NULL);
    hipEventSynchronize(stop); hipEventElapsedTime(&durationB, start, stop);
    checkCudaErrors(hipEventDestroy(start)); checkCudaErrors(hipEventDestroy(stop));
    print_log("Seeding completed in %.0f + %.0f ms\n", durationA, durationB);
    if (abort) return false;
  
    for (u32 i = 0; i < NB; i++) hipMemset(indexesE[1+i], 0, indexesSize);

    qA = sizeA/NB;
    const size_t qB = sizeB/NB;
    qE = NX2 / NB;
    for (u32 i = NB; i--; ) {
      if (tp.expand == 0)
        Round<1, EDGES_A, uint2, EDGES_B/NB, uint2><<<tp.trim.blocks/NB, tp.trim.tpb>>>(0, *dipkeys, (const uint2 *)(bufferA+i*qA), (uint2 *)(bufferB+i*qB), indexesE[0]+i*qE, indexesE[1+i]); // to .632
      else if (tp.expand == 1)
        Round<1, EDGES_A,   u32, EDGES_B/NB, uint2><<<tp.trim.blocks/NB, tp.trim.tpb>>>(0, *dipkeys, (const   u32 *)(bufferA+i*qA), (uint2 *)(bufferB+i*qB), indexesE[0]+i*qE, indexesE[1+i]); // to .632
      else // tp.expand == 2
        Round<1, EDGES_A,   u32, EDGES_B/NB,   u32><<<tp.trim.blocks/NB, tp.trim.tpb>>>(0, *dipkeys, (const   u32 *)(bufferA+i*qA), (  u32 *)(bufferB+i*qB), indexesE[0]+i*qE, indexesE[1+i]); // to .632
      if (abort) return false;
    }

    hipMemset(indexesE[0], 0, indexesSize);

    if (tp.expand < 2)
      Round<NB, EDGES_B/NB, uint2, EDGES_B/2, uint2><<<tp.trim.blocks, tp.trim.tpb>>>(1, *dipkeys, (const uint2 *)bufferB, (uint2 *)bufferA, indexesE[1], indexesE[0]); // to .296
    else
      Round<NB, EDGES_B/NB,   u32, EDGES_B/2, uint2><<<tp.trim.blocks, tp.trim.tpb>>>(1, *dipkeys, (const   u32 *)bufferB, (uint2 *)bufferA, indexesE[1], indexesE[0]); // to .296
    if (abort) return false;

    hipMemset(indexesE[1], 0, indexesSize);

    Round<1, EDGES_B/2, uint2, EDGES_A/4, uint2><<<tp.trim.blocks, tp.trim.tpb>>>(2, *dipkeys, (const uint2 *)bufferA, (uint2 *)bufferB, indexesE[0], indexesE[1]); // to .176
    if (abort) return false;

    hipMemset(indexesE[0], 0, indexesSize);

    Round<1, EDGES_A/4, uint2, EDGES_B/4, uint2><<<tp.trim.blocks, tp.trim.tpb>>>(3, *dipkeys, (const uint2 *)bufferB, (uint2 *)bufferA, indexesE[1], indexesE[0]); // to .117
    if (abort) return false;
  
    hipDeviceSynchronize();
  
    for (int round = 4; round < tp.ntrims; round += 2) {
      hipMemset(indexesE[1], 0, indexesSize);
      Round<1, EDGES_B/4, uint2, EDGES_B/4, uint2><<<tp.trim.blocks, tp.trim.tpb>>>(round, *dipkeys,  (const uint2 *)bufferA, (uint2 *)bufferB, indexesE[0], indexesE[1]);
      if (abort) return false;
      hipMemset(indexesE[0], 0, indexesSize);
      Round<1, EDGES_B/4, uint2, EDGES_B/4, uint2><<<tp.trim.blocks, tp.trim.tpb>>>(round+1, *dipkeys,  (const uint2 *)bufferB, (uint2 *)bufferA, indexesE[1], indexesE[0]);
      if (abort) return false;
    }
    
    hipMemset(indexesE[1], 0, indexesSize);
    hipDeviceSynchronize();
  
    Tail<EDGES_B/4><<<tp.tail.blocks, tp.tail.tpb>>>((const uint2 *)bufferA, (uint2 *)bufferB, indexesE[0], indexesE[1]);
    hipMemcpy(&nedges, indexesE[1], sizeof(u32), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    return nedges;
  }
};

#define IDXSHIFT 10
#define CUCKOO_SIZE (NNODES >> IDXSHIFT)
#define CUCKOO_MASK (CUCKOO_SIZE - 1)
// number of (least significant) key bits that survives leftshift by NODEBITS
#define KEYBITS (64-NODEBITS)
#define KEYMASK ((1L << KEYBITS) - 1)
#define MAXDRIFT (1L << (KEYBITS - IDXSHIFT))

class cuckoo_hash {
public:
  u64 *cuckoo;

  cuckoo_hash() {
    cuckoo = new u64[CUCKOO_SIZE];
  }
  ~cuckoo_hash() {
    delete[] cuckoo;
  }
  void set(node_t u, node_t v) {
    u64 niew = (u64)u << NODEBITS | v;
    for (node_t ui = u >> IDXSHIFT; ; ui = (ui+1) & CUCKOO_MASK) {
      u64 old = cuckoo[ui];
      if (old == 0 || (old >> NODEBITS) == (u & KEYMASK)) {
        cuckoo[ui] = niew;
        return;
      }
    }
  }
  node_t operator[](node_t u) const {
    for (node_t ui = u >> IDXSHIFT; ; ui = (ui+1) & CUCKOO_MASK) {
      u64 cu = cuckoo[ui];
      if (!cu)
        return 0;
      if ((cu >> NODEBITS) == (u & KEYMASK)) {
        assert(((ui - (u >> IDXSHIFT)) & CUCKOO_MASK) < MAXDRIFT);
        return (node_t)(cu & NODEMASK);
      }
    }
  }
};

const u32 MAXPATHLEN = 8 << ((NODEBITS+2)/3);

int nonce_cmp(const void *a, const void *b) {
  return *(u32 *)a - *(u32 *)b;
}

const u32 MAXEDGES = 0x20000;

struct solver_ctx {
  edgetrimmer trimmer;
  bool mutatenonce;
  uint2 *edges;
  cuckoo_hash *cuckoo;
  uint2 soledges[PROOFSIZE];
  std::vector<u32> sols; // concatenation of all proof's indices
  u32 us[MAXPATHLEN];
  u32 vs[MAXPATHLEN];

  solver_ctx(const trimparams tp, bool mutate_nonce) : trimmer(tp) {
    edges   = new uint2[MAXEDGES];
    cuckoo  = new cuckoo_hash();
    mutatenonce = mutate_nonce;
  }

  void setheadernonce(char * const headernonce, const u32 len, const u32 nonce) {
    if (mutatenonce) {
      ((u32 *)headernonce)[len/sizeof(u32)-1] = htole32(nonce); // place nonce at end
    }
    setheader(headernonce, len, &trimmer.sipkeys);
    sols.clear();
  }
  ~solver_ctx() {
    delete cuckoo;
    delete[] edges;
  }

  void recordedge(const u32 i, const u32 u2, const u32 v2) {
    soledges[i].x = u2/2;
    soledges[i].y = v2/2;
  }

  void solution(const u32 *us, u32 nu, const u32 *vs, u32 nv) {
    u32 ni = 0;
    recordedge(ni++, *us, *vs);
    while (nu--)
      recordedge(ni++, us[(nu+1)&~1], us[nu|1]); // u's in even position; v's in odd
    while (nv--)
    recordedge(ni++, vs[nv|1], vs[(nv+1)&~1]); // u's in odd position; v's in even
    assert(ni == PROOFSIZE);
    sols.resize(sols.size() + PROOFSIZE);
    hipMemcpyToSymbol(HIP_SYMBOL(recoveredges), soledges, sizeof(soledges));
    hipMemset(trimmer.indexesE[1], 0, trimmer.indexesSize);
    Recovery<<<trimmer.tp.recover.blocks, trimmer.tp.recover.tpb>>>(*trimmer.dipkeys, (ulonglong4*)trimmer.bufferA, (int *)trimmer.indexesE[1]);
    hipMemcpy(&sols[sols.size()-PROOFSIZE], trimmer.indexesE[1], PROOFSIZE * sizeof(u32), hipMemcpyDeviceToHost);
    checkCudaErrors_V(hipDeviceSynchronize());
    qsort(&sols[sols.size()-PROOFSIZE], PROOFSIZE, sizeof(u32), nonce_cmp);
  }

  u32 path(u32 u, u32 *us) {
    u32 nu, u0 = u;
    for (nu = 0; u; u = (*cuckoo)[u]) {
      if (nu >= MAXPATHLEN) {
        while (nu-- && us[nu] != u) ;
        if (~nu) {
          print_log("illegal %4d-cycle from node %d\n", MAXPATHLEN-nu, u0);
          exit(0);
        }
        print_log("maximum path length exceeded\n");
        return 0; // happens once in a million runs or so; signal trouble
      }
      us[nu++] = u;
    }
    return nu;
  }

  void addedge(uint2 edge) {
    const u32 u0 = edge.x << 1, v0 = (edge.y << 1) | 1;
    if (u0) {
      u32 nu = path(u0, us), nv = path(v0, vs);
      if (!nu-- || !nv--)
        return; // drop edge causing trouble
      // print_log("vx %02x ux %02x e %08x uxyz %06x vxyz %06x u0 %x v0 %x nu %d nv %d\n", vx, ux, e, uxyz, vxyz, u0, v0, nu, nv);
      if (us[nu] == vs[nv]) {
        const u32 min = nu < nv ? nu : nv;
        for (nu -= min, nv -= min; us[nu] != vs[nv]; nu++, nv++) ;
        const u32 len = nu + nv + 1;
        print_log("%4d-cycle found\n", len);
        if (len == PROOFSIZE)
          solution(us, nu, vs, nv);
        // if (len == 2) print_log("edge %x %x\n", edge.x, edge.y);
      } else if (nu < nv) {
        while (nu--)
          cuckoo->set(us[nu+1], us[nu]);
        cuckoo->set(u0, v0);
      } else {
        while (nv--)
          cuckoo->set(vs[nv+1], vs[nv]);
        cuckoo->set(v0, u0);
      }
    }
  }

  void findcycles(uint2 *edges, u32 nedges) {
    memset(cuckoo->cuckoo, 0, CUCKOO_SIZE * sizeof(u64));
    for (u32 i = 0; i < nedges; i++)
      addedge(edges[i]);
  }

  int solve() {
    u64 time0, time1;
    u32 timems,timems2;

    trimmer.abort = false;
    time0 = timestamp();
    u32 nedges = trimmer.trim();
    if (!nedges)
      return 0;
    if (nedges > MAXEDGES) {
      print_log("OOPS; losing %d edges beyond MAXEDGES=%d\n", nedges-MAXEDGES, MAXEDGES);
      nedges = MAXEDGES;
    }
    hipMemcpy(edges, trimmer.bufferB, sizeof(uint2[nedges]), hipMemcpyDeviceToHost);
    time1 = timestamp(); timems  = (time1 - time0) / 1000000;
    time0 = timestamp();
    findcycles(edges, nedges);
    time1 = timestamp(); timems2 = (time1 - time0) / 1000000;
    print_log("findcycles edges %d time %d ms total %d ms\n", nedges, timems2, timems+timems2);
    return sols.size() / PROOFSIZE;
  }

  void abort() {
    trimmer.abort = true;
  }
};

#include <unistd.h>

// arbitrary length of header hashed into siphash key
#define HEADERLEN 80

typedef solver_ctx SolverCtx;

CALL_CONVENTION int run_solver(SolverCtx* ctx,
                               char* header,
                               int header_length,
                               u32 nonce,
                               u32 range,
                               SolverSolutions *solutions,
                               SolverStats *stats
                               )
{
  u64 time0, time1;
  u32 timems;
  u32 sumnsols = 0;
  int device_id;
  if (stats != NULL) {
    hipGetDevice(&device_id);
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, stats->device_id);
    stats->device_id = device_id;
    stats->edge_bits = EDGEBITS;
    strncpy(stats->device_name, props.name, MAX_NAME_LEN);
  }

  if (ctx == NULL || !ctx->trimmer.initsuccess){
    print_log("Error initialising trimmer. Aborting.\n");
    print_log("Reason: %s\n", LAST_ERROR_REASON);
    if (stats != NULL) {
       stats->has_errored = true;
       strncpy(stats->error_reason, LAST_ERROR_REASON, MAX_NAME_LEN);
    }
    return 0;
  }

  for (u32 r = 0; r < range; r++) {
    time0 = timestamp();
    ctx->setheadernonce(header, header_length, nonce + r);
    print_log("nonce %d k0 k1 k2 k3 %llx %llx %llx %llx\n", nonce+r, ctx->trimmer.sipkeys.k0, ctx->trimmer.sipkeys.k1, ctx->trimmer.sipkeys.k2, ctx->trimmer.sipkeys.k3);
    u32 nsols = ctx->solve();
    time1 = timestamp();
    timems = (time1 - time0) / 1000000;
    print_log("Time: %d ms\n", timems);
    for (unsigned s = 0; s < nsols; s++) {
      print_log("Solution");
      u32* prf = &ctx->sols[s * PROOFSIZE];
      for (u32 i = 0; i < PROOFSIZE; i++)
        print_log(" %jx", (uintmax_t)prf[i]);
      print_log("\n");
      if (solutions != NULL){
        solutions->edge_bits = EDGEBITS;
        solutions->num_sols++;
        solutions->sols[sumnsols+s].nonce = nonce + r;
        for (u32 i = 0; i < PROOFSIZE; i++) 
          solutions->sols[sumnsols+s].proof[i] = (u64) prf[i];
      }
      int pow_rc = verify(prf, &ctx->trimmer.sipkeys);
      if (pow_rc == POW_OK) {
        print_log("Verified with cyclehash ");
        unsigned char cyclehash[32];
        blake2b((void *)cyclehash, sizeof(cyclehash), (const void *)prf, sizeof(proof), 0, 0);
        for (int i=0; i<32; i++)
          print_log("%02x", cyclehash[i]);
        print_log("\n");
      } else {
        print_log("FAILED due to %s\n", errstr[pow_rc]);
      }
    }
    sumnsols += nsols;
    if (stats != NULL) {
      stats->last_start_time = time0;
      stats->last_end_time = time1;
      stats->last_solution_time = time1 - time0;
    }
  }
  print_log("%d total solutions\n", sumnsols);
  return sumnsols > 0;
}

CALL_CONVENTION SolverCtx* create_solver_ctx(SolverParams* params) {
  trimparams tp;
  tp.ntrims = params->ntrims;
  tp.expand = params->expand;
  tp.genA.blocks = params->genablocks;
  tp.genA.tpb = params->genatpb;
  tp.genB.tpb = params->genbtpb;
  tp.trim.tpb = params->trimtpb;
  tp.tail.tpb = params->tailtpb;
  tp.recover.blocks = params->recoverblocks;
  tp.recover.tpb = params->recovertpb;

  hipDeviceProp_t prop;
  checkCudaErrors_N(hipGetDeviceProperties(&prop, params->device));

  assert(tp.genA.tpb <= prop.maxThreadsPerBlock);
  assert(tp.genB.tpb <= prop.maxThreadsPerBlock);
  assert(tp.trim.tpb <= prop.maxThreadsPerBlock);
  // assert(tp.tailblocks <= prop.threadDims[0]);
  assert(tp.tail.tpb <= prop.maxThreadsPerBlock);
  assert(tp.recover.tpb <= prop.maxThreadsPerBlock);

  assert(tp.genA.blocks * tp.genA.tpb <= NEDGES); // check THREADS_HAVE_EDGES
  assert(tp.recover.blocks * tp.recover.tpb <= NEDGES); // check THREADS_HAVE_EDGES
  assert(tp.genA.tpb / NX <= FLUSHA); // check ROWS_LIMIT_LOSSES
  assert(tp.genA.tpb / NX <= FLUSHA); // check COLS_LIMIT_LOSSES

  hipSetDevice(params->device);
  if (!params->cpuload)
    checkCudaErrors_N(hipSetDeviceFlags(hipDeviceScheduleBlockingSync));

  SolverCtx* ctx = new SolverCtx(tp, params->mutate_nonce);

  return ctx;
}

CALL_CONVENTION void destroy_solver_ctx(SolverCtx* ctx) {
  delete ctx;
}

CALL_CONVENTION void stop_solver(SolverCtx* ctx) {
  ctx->abort();
}

CALL_CONVENTION void fill_default_params(SolverParams* params) {
  trimparams tp;
  params->device = 0;
  params->ntrims = tp.ntrims;
  params->expand = tp.expand;
  params->genablocks = min(tp.genA.blocks, NEDGES/tp.genA.tpb);
  params->genatpb = tp.genA.tpb;
  params->genbtpb = tp.genB.tpb;
  params->trimtpb = tp.trim.tpb;
  params->tailtpb = tp.tail.tpb;
  params->recoverblocks = min(tp.recover.blocks, NEDGES/tp.recover.tpb);
  params->recovertpb = tp.recover.tpb;
  params->cpuload = false;
}

int main(int argc, char **argv) {
  trimparams tp;
  u32 nonce = 0;
  u32 range = 1;
  u32 device = 0;
  char header[HEADERLEN];
  u32 len;
  int c;

  // set defaults
  SolverParams params;
  fill_default_params(&params);

  memset(header, 0, sizeof(header));
  while ((c = getopt(argc, argv, "scb:d:E:h:k:m:n:r:U:u:v:w:y:Z:z:")) != -1) {
    switch (c) {
      case 's':
        print_log("SYNOPSIS\n  cuda%d [-s] [-c] [-d device] [-E 0-2] [-h hexheader] [-m trims] [-n nonce] [-r range] [-U seedAblocks] [-u seedAthreads] [-v seedBthreads] [-w Trimthreads] [-y Tailthreads] [-Z recoverblocks] [-z recoverthreads]\n", EDGEBITS);
        print_log("DEFAULTS\n  cuda%d -d %d -E %d -h \"\" -m %d -n %d -r %d -U %d -u %d -v %d -w %d -y %d -Z %d -z %d\n", EDGEBITS, device, tp.expand, tp.ntrims, nonce, range, tp.genA.blocks, tp.genA.tpb, tp.genB.tpb, tp.trim.tpb, tp.tail.tpb, tp.recover.blocks, tp.recover.tpb);
        exit(0);
      case 'c':
        params.cpuload = false;
        break;
      case 'd':
        device = params.device = atoi(optarg);
        break;
      case 'E':
        params.expand = atoi(optarg);
        assert(params.expand <= 2);
        break;
      case 'h':
        len = strlen(optarg)/2;
        assert(len <= sizeof(header));
        for (u32 i=0; i<len; i++)
          sscanf(optarg+2*i, "%2hhx", header+i); // hh specifies storage of a single byte
        break;
      case 'n':
        nonce = atoi(optarg);
        break;
      case 'm':
        params.ntrims = atoi(optarg) & -2; // make even as required by solve()
        break;
      case 'r':
        range = atoi(optarg);
        break;
      case 'U':
        params.genablocks = atoi(optarg);
        break;
      case 'u':
        params.genatpb = atoi(optarg);
        break;
      case 'v':
        params.genbtpb = atoi(optarg);
        break;
      case 'w':
        params.trimtpb = atoi(optarg);
        break;
      case 'y':
        params.tailtpb = atoi(optarg);
        break;
      case 'Z':
        params.recoverblocks = atoi(optarg);
        break;
      case 'z':
        params.recovertpb = atoi(optarg);
        break;
    }
  }
  int nDevices;
  checkCudaErrors(hipGetDeviceCount(&nDevices));
  assert(device < nDevices);
  hipDeviceProp_t prop;
  checkCudaErrors(hipGetDeviceProperties(&prop, device));
  u64 dbytes = prop.totalGlobalMem;
  int dunit;
  for (dunit=0; dbytes >= 102400; dbytes>>=10,dunit++) ;
  print_log("%s with %d%cB @ %d bits x %dMHz\n", prop.name, (u32)dbytes, " KMGT"[dunit], prop.memoryBusWidth, prop.memoryClockRate/1000);

  print_log("Looking for %d-cycle on cuckoo%d(\"%s\",%d", PROOFSIZE, EDGEBITS, header, nonce);
  if (range > 1)
    print_log("-%d", nonce+range-1);
  print_log(") with 50%% edges, %d*%d buckets, %d trims, and %d thread blocks.\n", NX, NY, params.ntrims, NX);

  SolverCtx* ctx = create_solver_ctx(&params);

  u64 bytes = ctx->trimmer.globalbytes();
  int unit;
  for (unit=0; bytes >= 102400; bytes>>=10,unit++) ;
  print_log("Using %d%cB of global memory.\n", (u32)bytes, " KMGT"[unit]);

  run_solver(ctx, header, sizeof(header), nonce, range, NULL, NULL);

  return 0;
}
