#include "hip/hip_runtime.h"
// Cuckatoo Cycle, a memory-hard proof-of-work by John Tromp
// Copyright (c) 2018-2019 Jiri Vadura (photon) and John Tromp
// This software is covered by the FAIR MINING license

#include <stdio.h>
#include <string.h>
#include <vector>
#include <assert.h>
#include "cuckatoo.h"
#include "graph.hpp"
#include "../crypto/siphash.cuh"
#include "../crypto/blake2.h"

typedef uint8_t u8;
typedef uint16_t u16;

#ifndef NA
#define NA 4
#endif
#define NA2 (NA * NA)

#ifndef MAXSOLS
#define MAXSOLS 4
#endif

#ifndef IDXSHIFT
// number of bits of compression of surviving edge endpoints
// reduces space used in cycle finding, but too high a value
// results in NODE OVERFLOW warnings and fake cycles
#define IDXSHIFT 12
#endif

const u32 MAXEDGES = NEDGES >> IDXSHIFT;

typedef uint64_t u64; // save some typing

#ifndef XBITS
// assumes at least 2^18 bits of shared mem (32 KB) on thread block
// #define XBITS ((EDGEBITS-18+1)/2)
// scrap that; too few buckets inhibits parallellism
#define XBITS 6
#endif

const u32 NX        = 1 << XBITS;
const u32 NX2       = NX * NX;
const u32 NX2_NA    = NX2 / NA;
const u32 YBITS     = XBITS;
const u32 NY        = 1 << YBITS;
const u32 YZBITS    = EDGEBITS - XBITS;
const u32 ZBITS     = YZBITS - YBITS;
const u32 NZ        = 1 << ZBITS;
const u32 ZMASK     = NZ - 1;

#ifndef NEPS_A
#define NEPS_A 133
#endif
#ifndef NEPS_B
#define NEPS_B 85
#endif
#define NEPS 128

const u32 EDGES_A = NZ * NEPS_A / NEPS;
const u32 EDGES_B = NZ * NEPS_B / NEPS;

const u32 ROW_EDGES_A = EDGES_A * NY;
const u32 ROW_EDGES_B = EDGES_B * NY;

// Number of rows in bufferB not overlapping bufferA
#ifndef NRB1
#define NRB1 (NX / 2)
#endif
#define NRB2 (NX - NRB1)
#define NB 2

__constant__ uint2 recoveredges[PROOFSIZE];
__constant__ siphash_keys dipkeys;

#ifndef FLUSHA // should perhaps be in trimparams and passed as template parameter
#define FLUSHA 4
#endif

#ifndef NFLUSH
#define NFLUSH (8 / FLUSHA)
#endif

__device__ __forceinline__  void bitmapset(u32 *ebitmap, const int bucket) {
  int word = bucket >> 5;
  unsigned char bit = bucket & 0x1F;
  const u32 mask = 1 << bit;
  atomicOr(ebitmap + word, mask);
}

__device__ __forceinline__  bool bitmaptest(u32 *ebitmap, const int bucket) {
  int word = bucket >> 5;
  unsigned char bit = bucket & 0x1F;
  return (ebitmap[word] >> bit) & 1;
}

__device__ __forceinline__  u32 endpoint(u32 nonce, bool uorv) {
  return dipnode(dipkeys, nonce, uorv);
}

__device__ __forceinline__  u32 endpoint(uint2 nodes, bool uorv) {
  return uorv ? nodes.y : nodes.x;
}

template<int tpb, int maxOut>
__global__ void Seed(u32 * __restrict__ buffer, u32 * __restrict__ indexes, const u64 offset) {
  const int group = blockIdx.x;
  const int dim = blockDim.x;
  const int lid = threadIdx.x;
  const int gid = group * dim + lid;
  const int nthreads = gridDim.x * dim;

  extern __shared__ u32 tmp[][FLUSHA-1];
  int *counters = (int *)(tmp[NX2]);

#if tpb && NX2 % tpb == 0
  for (int i = 0; i < NX2/tpb; i++)
#else
  for (int i = 0; i < (NX2 - lid + tpb-1) / tpb; i++)
#endif
    counters[lid + tpb * i] = 0;

  __syncthreads();

  const int nloops = (NEDGES / NA - gid + nthreads-1) / nthreads;
  for (int i = 0; i < nloops; i++) {
    const u32 nonce = offset + gid * nloops + i;
    const u32 node0 = endpoint(nonce, 0);
    const int bucket = node0 >> ZBITS;
    u32 counter = (int)atomicAdd(counters + bucket, 1);
    for (int nf=0; ; nf++) {
      if (counter < FLUSHA-1)
        tmp[bucket][counter] = nonce;
      __syncthreads();
      if (nf == NFLUSH)
        break;
      if (counter == FLUSHA-1) {
        const u64 pos = min((int)atomicAdd(indexes + bucket, FLUSHA), (int)(maxOut - FLUSHA));
        const u64 idx = (bucket + (bucket / NX2_NA) * (NX2 - NX2_NA)) * maxOut + pos;
#if FLUSHA==4
        ((uint4 *)buffer)[idx/4] = make_uint4(tmp[bucket][0], tmp[bucket][1], tmp[bucket][2], nonce);
#elif FLUSHA==2
        ((uint2 *)buffer)[idx/2] = make_uint2(tmp[bucket][0], nonce);
#endif
        counters[bucket] %= FLUSHA;
      }
      __syncthreads();
      counter -= FLUSHA;
    }
    if ((int)counter >= FLUSHA-1) printf("WHOOPS!\n");
  }
  for (int i = 0; i < NX2 / tpb; i++) {
    const int bucket = lid + i * tpb;
    const int cnt = counters[bucket];
    if (cnt) {
      const u64 pos = min((int)atomicAdd(indexes + bucket, FLUSHA), (int)(maxOut - FLUSHA));
      const u64 idx = (bucket + (bucket / NX2_NA) * (NX2 - NX2_NA)) * maxOut + pos;
#if FLUSHA==4
        ((uint4 *)buffer)[idx/4] = make_uint4(tmp[bucket][0], cnt >= 2 ? tmp[bucket][1] : 0, cnt >= 3 ? tmp[bucket][2] : 0, 0);
#elif FLUSHA==2
        ((uint2 *)buffer)[idx/2] = make_uint2(tmp[bucket][0], 0);
#endif
    }
  }
}

#ifndef PART_BITS
// #bits used to partition edge set processing to save shared memory
// a value of 0 does no partitioning and is fastest
// a value of 1 partitions in two at about 33% slowdown
// higher values are not that interesting
#define PART_BITS 0
#endif

const u32 PART_MASK = (1 << PART_BITS) - 1;
const u32 NONPART_BITS = ZBITS - PART_BITS;
const word_t NONPART_MASK = (1 << NONPART_BITS) - 1;
const int BITMAPBYTES = (NZ >> PART_BITS) / 8;

template<int tpb, int maxIn, int maxOut>
__global__ void Round0(const int part, u32 * src, u32 * dst, u32 * srcIdx, u32 * dstIdx, const int offset) {
  const int group = blockIdx.x;
  const int lid = threadIdx.x;
  const int BITMAPWORDS = BITMAPBYTES / sizeof(u32);
  int nloops[NA];

  extern __shared__ u32 ebitmap[];

#if tpb && BITMAPWORDS% tpb == 0
  for (int i = 0; i < BITMAPWORDS/tpb; i++)
#else
  for (int i = 0; i < (BITMAPWORDS- lid + tpb-1) / tpb; i++)
#endif
    ebitmap[lid + tpb * i] = 0;

  for (int a = 0; a < NA; a++)
    nloops[a] = (min(srcIdx[a * NX2 + offset + group], maxIn) - lid + tpb-1) / tpb;

  const int rowOffset = offset * NA;
  src += maxIn * (rowOffset + group) + lid;

  __syncthreads();

  for (int a = 0; a < NA; a++) {
    const int delta = a * NX2_NA * maxIn;
    for (int i = 0; i < nloops[a]; i++) {
      u32 edge = src[delta + i * tpb];
      if (!edge) continue;
      u32 z = endpoint(edge, 0) & ZMASK;
      if ((z >> NONPART_BITS) == part)
        bitmapset(ebitmap, z & NONPART_MASK);
    }
  }

  __syncthreads();

  for (int a = 0; a < NA; a++) {
    const int delta = a * NX2_NA * maxIn;
    for (int i = 0; i < nloops[a]; i++) {
      const u32 edge = src[delta + i * tpb];
      if (!edge) continue;
      const u32 node0 = endpoint(edge, 0);
      const u32 z = node0 & ZMASK;
      if ((z >> NONPART_BITS) == part && bitmaptest(ebitmap, (z & NONPART_MASK) ^ 1)) {
        const int bucket = endpoint(edge, 1) >> ZBITS;
        const int bktIdx = min(atomicAdd(dstIdx + bucket + rowOffset, 1), maxOut - 1);
        dst[bucket * maxOut + bktIdx] = edge;
      }
    }
  }
}

template<int tpb, int maxIn, int maxOut>
__global__ void Round1(const int part, u32 * src, u32 * dst, u32 * srcIdx, u32 * dstIdx) {
  const int group = blockIdx.x;
  const int lid = threadIdx.x;

  const int BITMAPWORDS = BITMAPBYTES / sizeof(u32);
  int nloops[NA];

  extern __shared__ u32 ebitmap[];

#if tpb && BITMAPWORDS% tpb == 0
  for (int i = 0; i < BITMAPWORDS/tpb; i++)
#else
  for (int i = 0; i < (BITMAPWORDS- lid + tpb-1) / tpb; i++)
#endif
    ebitmap[lid + tpb * i] = 0;

  for (int a = 0; a < NA; a++)
    nloops[a] = (min(srcIdx[a * NX2 + group], maxIn) - lid + tpb-1) / tpb;

  src += maxIn * group + lid;

  __syncthreads();

  for (int a = 0; a < NA; a++) {
    const int delta = a * maxIn * NX2;
    for (int i = 0; i < nloops[a]; i++) {
      u32 edge = src[delta + i * tpb];
      u32 z = endpoint(edge, 1) & ZMASK;
      if ((z >> NONPART_BITS) == part)
        bitmapset(ebitmap, z & NONPART_MASK);
    }
  }

  __syncthreads();

  for (int a = 0; a < NA; a++) {
    const int delta = a * maxIn * NX2;
    for (int i = 0; i < nloops[a]; i++) {
      const u32 edge = src[delta + i * tpb];
      const u32 node1 = endpoint(edge, 1);
      const u32 z = node1 & ZMASK;
      if ((z >> NONPART_BITS) == part && bitmaptest(ebitmap, (z & NONPART_MASK) ^ 1)) {
        const int bucket = endpoint(edge, 0) >> ZBITS;
        const int bktIdx = min(atomicAdd(dstIdx + bucket, 1), maxOut - 1);
        dst[bucket * maxOut + bktIdx] = edge;
      }
    }
  }
}

template<int tpb, int maxIn, typename EdgeIn, int maxOut>
__global__ void Round(const int round, const int part, EdgeIn * src, uint2 * dst, u32 * srcIdx, u32 * dstIdx) {
  const int lid = threadIdx.x;
  const int group = blockIdx.x;

  const int BITMAPWORDS = BITMAPBYTES / sizeof(u32);

  extern __shared__ u32 ebitmap[];

  const int nloops = (min(srcIdx[group], maxIn) - lid + tpb-1) / tpb;

#if tpb && BITMAPWORDS % tpb == 0
  for (int i = 0; i < BITMAPWORDS/tpb; i++)
#else
  for (int i = 0; i < (BITMAPWORDS - lid + tpb-1) / tpb; i++)
#endif
    ebitmap[lid + tpb * i] = 0;

  src += maxIn * group + lid;

  __syncthreads();

  for (int i = 0; i < nloops; i++) {
    const EdgeIn edge = src[i * tpb]; // EdgeIn edge = __ldg(&src[index]);
    const u32 z = endpoint(edge, 0) & ZMASK;
    if ((z >> NONPART_BITS) == part)
      bitmapset(ebitmap, z & NONPART_MASK);
  }

  __syncthreads();

  for (int i = 0; i < nloops; i++) {
    const EdgeIn edge = src[i * tpb]; // EdgeIn edge = __ldg(&src[index]);
    const u32 node = endpoint(edge, 0);
    const u32 z = node & ZMASK;
    if ((z >> NONPART_BITS) == part && bitmaptest(ebitmap, (z & NONPART_MASK) ^ 1)) {
      u32 node2 = endpoint(edge, 1);
      const int bucket = node2 >> ZBITS;
      const int bktIdx = min(atomicAdd(dstIdx + bucket, 1), maxOut - 1);
      dst[bucket * maxOut + bktIdx] = make_uint2(node2, node);
    }
  }
}

#ifndef LISTBITS
#define LISTBITS 12
#endif

const u32 NLISTS  = 1 << LISTBITS;
const u32 LISTMASK = NLISTS - 1;

#ifndef NNEXTS
#define NNEXTS NLISTS
#endif

template<int tpb, int bktInSize, int bktOutSize>
__global__  void Relay(const u32 round, const uint2 * source, uint2 * destination, const u32 * sourceIndexes, u32 * destinationIndexes, bool TAGGED)
{
  const int lid = threadIdx.x;
  const int group = blockIdx.x;

  __shared__ u32 lists[NLISTS];
  __shared__ u32 nexts[NNEXTS];

  const int nloops = (min(sourceIndexes[group], NNEXTS) - lid + tpb-1) / tpb;

  source += bktInSize * group;

  for (int i = 0; i < NLISTS/tpb; i++)
    lists[i * tpb + lid] = ~0;

  __syncthreads();

  for (int i = 0; i < nloops; i++) {
    const u32 index = i * tpb + lid;
    const u32 list = endpoint(source[index], 0) & LISTMASK;
    nexts[index] = atomicExch(&lists[list], index);
  }

  __syncthreads();

  for (int i = nloops; --i >= 0;) {
    const u32 index = i * tpb + lid;
    const uint2 edge = source[index];
    if (edge.y & NEDGES) continue; // copies don't relay
    u32 bucket = edge.y >> ZBITS;
    u32 copybit = 0;
    const u32 list = (edge.x & LISTMASK) ^ 1;
    for (u32 idx = lists[list]; idx != ~0; idx = nexts[idx]) {
      uint2 tagged = source[idx];
      if ((tagged.x ^ edge.x ^ 1) & ZMASK) continue;
      u32 bktIdx = min(atomicAdd(destinationIndexes + bucket, 1), bktOutSize - 1);
      u32 tag = TAGGED ? tagged.x >> ZBITS : tagged.y >> 1;
      destination[(bucket * bktOutSize) + bktIdx] = make_uint2((tag << ZBITS) | (edge.y & ZMASK), copybit | (group << ZBITS) | (edge.x & ZMASK));
      copybit = NEDGES;
    }
  }
}

template<int tpb, int maxIn>
__global__ void Tail(const uint2 *source, uint4 *destination, const u32 *sourceIndexes, u32 *destinationIndexes) {
  const int lid = threadIdx.x;
  const int group = blockIdx.x;

  __shared__ u32 lists[NLISTS];
  __shared__ u32 nexts[NNEXTS];

  const int nloops = (min(sourceIndexes[group], NNEXTS) - lid + tpb-1) / tpb;

  source += maxIn * group;

  for (int i = 0; i < NLISTS/tpb; i++)
    lists[i * tpb + lid] = ~0;

  __syncthreads();

  for (int i = 0; i < nloops; i++) {
    const u32 index = i * tpb + lid;
    const u32 list = source[index].x & LISTMASK;
    nexts[index] = atomicExch(&lists[list], index);
  }

  __syncthreads();

  for (int i = nloops; --i >= 0;) {
    const u32 index = i * tpb + lid;
    const uint2 edge = source[index];
#ifdef DBG101
    if (((edge.x^0x1d3cc2ae)&ZMASK)<2) printf("Tail group %x x %x y %x tag %x\n", group, edge.x, edge.y, edge.x>>ZBITS);
    if (((edge.y^0x1d3cc2ae)&ZMASK)<2) printf("Tail group %x x %x y %x tag %x\n", group, edge.x, edge.y, edge.x>>ZBITS);
#endif
    if (edge.x & 1) continue;
    const u32 list = (edge.x & LISTMASK) ^ 1;
    for (u32 idx = lists[list]; idx != ~0; idx = nexts[idx]) {
      uint2 other = source[idx];
      if ((other.x ^ edge.x) != 1) continue;
      u32 bktIdx = atomicAdd(destinationIndexes, 2);
      destination[bktIdx/2] = make_uint4(edge.y & (NEDGES-1), (group << ZBITS) |  (edge.x & ZMASK),
                                        other.y & (NEDGES-1), (group << ZBITS) | (other.x & ZMASK));
    }
  }
}

#define checkCudaErrors_V(ans) ({if (gpuAssert((ans), __FILE__, __LINE__) != hipSuccess) return;})
#define checkCudaErrors_N(ans) ({if (gpuAssert((ans), __FILE__, __LINE__) != hipSuccess) return NULL;})
#define checkCudaErrors(ans) ({int retval = gpuAssert((ans), __FILE__, __LINE__); if (retval != hipSuccess) return retval;})

inline int gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
  int device_id;
  hipGetDevice(&device_id);
  if (code != hipSuccess) {
    snprintf(LAST_ERROR_REASON, MAX_NAME_LEN, "Device %d GPUassert: %s %s %d", device_id, hipGetErrorString(code), file, line);
    hipDeviceReset();
    if (abort) return code;
  }
  return code;
}

__global__ void Recovery(u32 *indexes) {
  const int gid = blockDim.x * blockIdx.x + threadIdx.x;
  const int lid = threadIdx.x;
  const int nthreads = blockDim.x * gridDim.x;
  __shared__ u32 nonces[PROOFSIZE];
  
  const int nloops = (NEDGES -gid + nthreads-1) / nthreads;
  if (lid < PROOFSIZE) nonces[lid] = 0;
  __syncthreads();
  for (int i = 0; i < nloops; i++) {
    u64 nonce = gid * nloops + i;
    u64 u = endpoint(nonce, 0);
    u64 v = endpoint(nonce, 1);
    for (int i = 0; i < PROOFSIZE; i++) {
      if (recoveredges[i].x == v && recoveredges[i].y == u)
        nonces[i] = nonce;
    }
  }
  __syncthreads();
  if (lid < PROOFSIZE) {
    if (nonces[lid] > 0)
      indexes[lid] = nonces[lid];
  }
}

struct blockstpb {
  u16 blocks;
  u16 tpb;
};

#ifndef SEED_TPB
#define SEED_TPB 256
#endif
#ifndef TRIM0_TPB
#define TRIM0_TPB 1024
#endif
#ifndef TRIM1_TPB
#define TRIM1_TPB 512
#endif
#ifndef TRIM_TPB
#define TRIM_TPB 512
#endif
#ifndef RELAY_TPB
#define RELAY_TPB 512
#endif
#ifndef TAIL_TPB
#define TAIL_TPB SEED_TPB
#endif

struct trimparams {
  u16 ntrims;
  blockstpb seed;
  blockstpb trim0;
  blockstpb trim1;
  blockstpb trim;
  blockstpb tail;
  blockstpb recover;

  trimparams() {
    ntrims         =        31;
    seed.blocks    =      1024;
    seed.tpb       =  SEED_TPB;
    trim0.blocks   =    NX2_NA;
    trim0.tpb      = TRIM0_TPB;
    trim1.blocks   =    NX2_NA;
    trim1.tpb      = TRIM1_TPB;
    trim.blocks    =       NX2;
    trim.tpb       =  TRIM_TPB;
    tail.blocks    =       NX2;
    tail.tpb       =  TAIL_TPB;;
    recover.blocks =      2048;
    recover.tpb    =       256;
  }
};

typedef u32 proof[PROOFSIZE];

// maintains set of trimmable edges
struct edgetrimmer {
  trimparams tp;
  edgetrimmer *dt;
  size_t sizeA, sizeB;
  const size_t indexesSize = NX * NY * sizeof(u32);
  const size_t indexesSizeNA = NA * indexesSize;
  u8 *bufferA;
  u8 *bufferB;
  u8 *bufferA1;
  u32 *indexesA;
  u32 *indexesB;
  u32 nedges;
  u32 *uvnodes;
  siphash_keys sipkeys;
  bool abort;
  bool initsuccess = false;

  edgetrimmer(const trimparams _tp) : tp(_tp) {
    checkCudaErrors_V(hipMalloc((void**)&dt, sizeof(edgetrimmer)));
    checkCudaErrors_V(hipMalloc((void**)&uvnodes, PROOFSIZE * 2 * sizeof(u32)));
    checkCudaErrors_V(hipMalloc((void**)&indexesA, indexesSizeNA));
    checkCudaErrors_V(hipMalloc((void**)&indexesB, indexesSizeNA));
    sizeA = (u64)ROW_EDGES_A * NX * sizeof(u32);
    sizeB = (u64)ROW_EDGES_B * NX * sizeof(u32);
    const size_t bufferSize = sizeA + sizeB / NA;
    checkCudaErrors_V(hipMalloc((void**)&bufferB, bufferSize));
    bufferA = bufferB + sizeB / NA;
    bufferA1 = bufferB + sizeB;
  print_log("allocated %lld bytes bufferB %llx bufferA %llx bufferA1 %llx\n", bufferSize, bufferB, bufferA, bufferA1);
  print_log("endB %llx endA %llx endBuffer %llx\n", bufferB+sizeB, bufferA+sizeA, bufferB+bufferSize);
    assert((NA & (NA-1)) == 0); // ensure NA is a 2 power
    hipMemcpy(dt, this, sizeof(edgetrimmer), hipMemcpyHostToDevice);
    initsuccess = true;
    int maxbytes = 0x10000; // 64 KB
    hipFuncSetAttribute(reinterpret_cast<const void*>(Seed  < SEED_TPB), EDGES_A/NA                 >, hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes);
    hipFuncSetAttribute(reinterpret_cast<const void*>(Round0<TRIM0_TPB), EDGES_A/NA,      EDGES_B/NA>, hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes);
    hipFuncSetAttribute(reinterpret_cast<const void*>(Round1<TRIM1_TPB), EDGES_B/NA,       EDGES_B/2>, hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes);
    hipFuncSetAttribute(reinterpret_cast<const void*>(Round < TRIM_TPB), EDGES_B/2,   u32, EDGES_A/4>, hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes);
    hipFuncSetAttribute(reinterpret_cast<const void*>(Round < TRIM_TPB), EDGES_A/4, uint2, EDGES_B/4>, hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes);
    hipFuncSetAttribute(reinterpret_cast<const void*>(Round < TRIM_TPB), EDGES_B/4, uint2, EDGES_B/4>, hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes);
  }
  u64 globalbytes() const {
    return sizeA + sizeB/NA + 2 * indexesSizeNA + sizeof(siphash_keys) + PROOFSIZE * 2*sizeof(u32) + sizeof(edgetrimmer);
  }
  ~edgetrimmer() {
    checkCudaErrors_V(hipFree(bufferB));
    checkCudaErrors_V(hipFree(indexesA));
    checkCudaErrors_V(hipFree(indexesB));
    checkCudaErrors_V(hipFree(uvnodes));
    checkCudaErrors_V(hipFree(dt));
    hipDeviceReset();
  }

#ifndef VBIDX
#define VBIDX 0
#endif

  void indexcount(u32 round, const u32 *indexes) {
#ifdef VERBOSE
    u32 nedges;
    hipMemcpy(&nedges, indexes+VBIDX, sizeof(u32), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    print_log("round %d edges %d\n", round, nedges);
#endif

  }
  u32 trim() {
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start)); checkCudaErrors(hipEventCreate(&stop));
    hipMemcpyToSymbol(HIP_SYMBOL(dipkeys), &sipkeys, sizeof(sipkeys));
  
    hipDeviceSynchronize();
    float durationA;
    hipEventRecord(start, NULL);
  
    hipMemset(indexesA, 0, indexesSizeNA);
    for (u32 i = 0; i < NA; i++) {
      Seed<SEED_TPB, EDGES_A/NA><<<tp.seed.blocks, SEED_TPB, BITMAPBYTES>>>((u32*)(bufferA+i*(sizeA/NA2)), indexesA+i*NX2, i*(NEDGES/NA));
    hipDeviceSynchronize();
      if (abort) return false;
    }
  
    checkCudaErrors(hipDeviceSynchronize()); hipEventRecord(stop, NULL);
    hipEventSynchronize(stop); hipEventElapsedTime(&durationA, start, stop);
    hipEventRecord(start, NULL);
  
    // checkCudaErrors(hipEventDestroy(start)); checkCudaErrors(hipEventDestroy(stop));
    print_log("Seeding completed in %.0f ms\n", durationA);
    if (abort) return false;
  
    hipMemset(indexesB, 0, indexesSizeNA);

    const size_t qB = sizeB / NA;
    for (u32 i = 0; i < NA; i++) {
      for (u32 part = 0; part <= PART_MASK; part++) {
        Round0<TRIM0_TPB, EDGES_A/NA, EDGES_B/NA><<<NX2_NA, TRIM0_TPB, BITMAPBYTES>>>(part, (u32*)bufferA, (u32*)(bufferB+i*qB), indexesA, indexesB, i*NX2_NA); // to .632
        if (abort) return false;
      }
    }
    indexcount(1, indexesB);

    hipMemset(indexesA, 0, indexesSize);

    for (u32 part = 0; part <= PART_MASK; part++) {
      Round1<TRIM1_TPB, EDGES_B/NA, EDGES_B/2><<<NX2, TRIM1_TPB, BITMAPBYTES>>>(part, (u32*)bufferB, (u32*)bufferA1, indexesB, indexesA); // to .296
      if (abort) return false;
    }
    indexcount(2, indexesA);

    hipMemset(indexesB, 0, indexesSize);

    for (u32 part = 0; part <= PART_MASK; part++) {
      Round<TRIM_TPB, EDGES_B/2, u32, EDGES_A/4><<<NX2, TRIM_TPB, BITMAPBYTES>>>(2, part, (u32 *)bufferA1, (uint2 *)bufferB, indexesA, indexesB); // to .176
      if (abort) return false;
    }
    indexcount(3, indexesB);

    hipMemset(indexesA, 0, indexesSize);

    for (u32 part = 0; part <= PART_MASK; part++) {
      Round<TRIM_TPB, EDGES_A/4, uint2, EDGES_B/4><<<NX2, TRIM_TPB, BITMAPBYTES>>>(3, part, (uint2 *)bufferB, (uint2 *)bufferA1, indexesB, indexesA); // to .116
      if (abort) return false;
    }
    indexcount(4, indexesA);
  
    for (int round = 5; round < tp.ntrims + PROOFSIZE/2-1; round += 2) {
      hipMemset(indexesB, 0, indexesSize);
      if (round >= tp.ntrims)
        Relay<RELAY_TPB, EDGES_B/4, EDGES_B/4><<<NX2, RELAY_TPB>>>(round-1, (uint2 *)bufferA1, (uint2 *)bufferB, indexesA, indexesB, round > tp.ntrims);
      else for (u32 part = 0; part <= PART_MASK; part++) {
        Round<TRIM_TPB, EDGES_B/4, uint2, EDGES_B/4><<<NX2, TRIM_TPB, BITMAPBYTES>>>(round-1, part, (uint2 *)bufferA1, (uint2 *)bufferB, indexesA, indexesB);
      }
      indexcount(round, indexesB);
      if (abort) return false;
      hipMemset(indexesA, 0, indexesSize);
      if (round+1 >= tp.ntrims)
        Relay<RELAY_TPB, EDGES_B/4, EDGES_B/4><<<NX2, RELAY_TPB>>>(round, (uint2 *)bufferB, (uint2 *)bufferA1, indexesB, indexesA, round+1 > tp.ntrims);
      else for (u32 part = 0; part <= PART_MASK; part++) {
        Round<TRIM_TPB, EDGES_B/4, uint2, EDGES_B/4><<<NX2, TRIM_TPB, BITMAPBYTES>>>(round, part, (uint2 *)bufferB, (uint2 *)bufferA1, indexesB, indexesA);
      }
      indexcount(round+1, indexesA);
      if (abort) return false;
    }
    
    hipMemset(indexesB, 0, indexesSize);
    hipDeviceSynchronize();
  
    Tail<TAIL_TPB, EDGES_B/4><<<NX2, TAIL_TPB>>>((const uint2 *)bufferA1, (uint4 *)bufferB, indexesA, indexesB);
    hipMemcpy(&nedges, indexesB, sizeof(u32), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    return nedges;
  }
};

struct solver_ctx {
  edgetrimmer trimmer;
  bool mutatenonce;
  uint2 *edges;
  graph<word_t> cg;
  uint2 soledges[PROOFSIZE];
  std::vector<u32> sols; // concatenation of all proof's indices

  solver_ctx(const trimparams tp, bool mutate_nonce) : trimmer(tp), cg(MAXEDGES, MAXEDGES, MAXSOLS, IDXSHIFT) {
    edges   = new uint2[MAXEDGES];
    mutatenonce = mutate_nonce;
  }

  void setheadernonce(char * const headernonce, const u32 len, const u32 nonce) {
    if (mutatenonce) {
      ((u32 *)headernonce)[len/sizeof(u32)-1] = htole32(nonce); // place nonce at end
    }
    setheader(headernonce, len, &trimmer.sipkeys);
    sols.clear();
  }
  ~solver_ctx() {
    delete[] edges;
  }

  u32 findcycles(uint2 *edges, u32 nedges) {
    u32 ndupes = 0;
    cg.reset();
    for (u32 i = 0; i < nedges; i++)
      ndupes += !cg.add_compress_edge(edges[i].x, edges[i].y);
    for (u32 s = 0 ;s < cg.nsols; s++) {
#ifdef VERBOSE
      print_log("Solution");
#endif
      for (u32 j = 0; j < PROOFSIZE; j++) {
        soledges[j] = edges[cg.sols[s][j]];
#ifdef VERBOSE
	print_log(" (%x, %x)", soledges[j].x, soledges[j].y);
#endif
      }
#ifdef VERBOSE
      print_log("\n");
#endif
      sols.resize(sols.size() + PROOFSIZE);
      hipMemcpyToSymbol(HIP_SYMBOL(recoveredges), soledges, sizeof(soledges));
      hipMemset(trimmer.indexesB, 0, trimmer.indexesSize);
      Recovery<<<trimmer.tp.recover.blocks, trimmer.tp.recover.tpb>>>((u32 *)trimmer.bufferA1);
      hipMemcpy(&sols[sols.size()-PROOFSIZE], trimmer.bufferA1, PROOFSIZE * sizeof(u32), hipMemcpyDeviceToHost);
      checkCudaErrors(hipDeviceSynchronize());
      qsort(&sols[sols.size()-PROOFSIZE], PROOFSIZE, sizeof(u32), cg.nonce_cmp);
    }
    return ndupes;
  }

  int solve() {
    u64 time0, time1;
    u32 timems,timems2;

    trimmer.abort = false;
    time0 = timestamp();
    u32 nedges = trimmer.trim();
    // if (!nedges)
      // return 0;
    if (nedges > MAXEDGES) {
      print_log("OOPS; losing %d edges beyond MAXEDGES=%d\n", nedges-MAXEDGES, MAXEDGES);
      nedges = MAXEDGES;
      return 0;
    }
    hipMemcpy(edges, trimmer.bufferB, nedges * 8, hipMemcpyDeviceToHost);
    time1 = timestamp(); timems  = (time1 - time0) / 1000000;
    time0 = timestamp();
    u32 ndupes = findcycles(edges, nedges);
    time1 = timestamp(); timems2 = (time1 - time0) / 1000000;
    print_log("%d trims %d ms %d edges %d dupes %d ms total %d ms\n", trimmer.tp.ntrims, timems, nedges, ndupes, timems2, timems+timems2);
    return sols.size() / PROOFSIZE;
  }

  void abort() {
    trimmer.abort = true;
  }

};

#include <unistd.h>

// arbitrary length of header hashed into siphash key
#define HEADERLEN 80

typedef solver_ctx SolverCtx;

CALL_CONVENTION int run_solver(SolverCtx* ctx,
                               char* header,
                               int header_length,
                               u32 nonce,
                               u32 range,
                               SolverSolutions *solutions,
                               SolverStats *stats
                               )
{
  u64 time0, time1;
  u32 timems;
  u32 sumnsols = 0;
  int device_id;
  if (stats != NULL) {
    hipGetDevice(&device_id);
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, stats->device_id);
    stats->device_id = device_id;
    stats->edge_bits = EDGEBITS;
    strncpy(stats->device_name, props.name, MAX_NAME_LEN);
  }

  if (ctx == NULL || !ctx->trimmer.initsuccess){
    print_log("Error initialising trimmer. Aborting.\n");
    print_log("Reason: %s\n", LAST_ERROR_REASON);
    if (stats != NULL) {
       stats->has_errored = true;
       strncpy(stats->error_reason, LAST_ERROR_REASON, MAX_NAME_LEN);
    }
    return 0;
  }

  for (u32 r = 0; r < range; r++) {
    time0 = timestamp();
    ctx->setheadernonce(header, header_length, nonce + r);
    print_log("nonce %d k0 k1 k2 k3 %llx %llx %llx %llx\n", nonce+r, ctx->trimmer.sipkeys.k0, ctx->trimmer.sipkeys.k1, ctx->trimmer.sipkeys.k2, ctx->trimmer.sipkeys.k3);
    u32 nsols = ctx->solve();
    time1 = timestamp();
    timems = (time1 - time0) / 1000000;
    print_log("Time: %d ms\n", timems);
    for (unsigned s = 0; s < nsols; s++) {
      print_log("Solution");
      u32* prf = &ctx->sols[s * PROOFSIZE];
      for (u32 i = 0; i < PROOFSIZE; i++)
        print_log(" %jx", (uintmax_t)prf[i]);
      print_log("\n");
      if (solutions != NULL){
        solutions->edge_bits = EDGEBITS;
        solutions->num_sols++;
        solutions->sols[sumnsols+s].nonce = nonce + r;
        for (u32 i = 0; i < PROOFSIZE; i++) 
          solutions->sols[sumnsols+s].proof[i] = (u64) prf[i];
      }
      int pow_rc = verify(prf, &ctx->trimmer.sipkeys);
      if (pow_rc == POW_OK) {
        print_log("Verified with cyclehash ");
        unsigned char cyclehash[32];
        blake2b((void *)cyclehash, sizeof(cyclehash), (const void *)prf, sizeof(proof), 0, 0);
        for (int i=0; i<32; i++)
          print_log("%02x", cyclehash[i]);
        print_log("\n");
      } else {
        print_log("FAILED due to %s\n", errstr[pow_rc]);
      }
    }
    sumnsols += nsols;
    if (stats != NULL) {
      stats->last_start_time = time0;
      stats->last_end_time = time1;
      stats->last_solution_time = time1 - time0;
    }
  }
  print_log("%d total solutions\n", sumnsols);
  return sumnsols > 0;
}

CALL_CONVENTION SolverCtx* create_solver_ctx(SolverParams* params) {
  trimparams tp;
  tp.ntrims = params->ntrims;
  tp.seed.blocks = params->genablocks;
  tp.seed.tpb = params->genatpb;
  tp.trim0.tpb = params->genbtpb;
  tp.trim.tpb = params->trimtpb;
  tp.tail.tpb = params->tailtpb;
  tp.recover.blocks = params->recoverblocks;
  tp.recover.tpb = params->recovertpb;

  hipDeviceProp_t prop;
  checkCudaErrors_N(hipGetDeviceProperties(&prop, params->device));

  assert(tp.seed.tpb <= prop.maxThreadsPerBlock);
  assert(tp.trim0.tpb <= prop.maxThreadsPerBlock);
  assert(tp.trim.tpb <= prop.maxThreadsPerBlock);
  // assert(tp.tailblocks <= prop.threadDims[0]);
  assert(tp.tail.tpb <= prop.maxThreadsPerBlock);
  assert(tp.recover.tpb <= prop.maxThreadsPerBlock);

  hipSetDevice(params->device);
  if (!params->cpuload)
    checkCudaErrors_N(hipSetDeviceFlags(hipDeviceScheduleBlockingSync));

  SolverCtx* ctx = new SolverCtx(tp, params->mutate_nonce);

  return ctx;
}

CALL_CONVENTION void destroy_solver_ctx(SolverCtx* ctx) {
  delete ctx;
}

CALL_CONVENTION void stop_solver(SolverCtx* ctx) {
  ctx->abort();
}

CALL_CONVENTION void fill_default_params(SolverParams* params) {
  trimparams tp;
  params->device = 0;
  params->ntrims = tp.ntrims;
  params->genablocks = tp.seed.blocks;
  params->genatpb = tp.seed.tpb;
  params->genbtpb = tp.trim0.tpb;
  params->trimtpb = tp.trim.tpb;
  params->tailtpb = tp.tail.tpb;
  params->recoverblocks = min(tp.recover.blocks, (u32)(NEDGES/tp.recover.tpb));
  params->recovertpb = tp.recover.tpb;
  params->cpuload = false;
}

int main(int argc, char **argv) {
  trimparams tp;
  u32 nonce = 0;
  u32 range = 1;
  u32 device = 0;
  char header[HEADERLEN];
  u32 len;
  int c;

  // set defaults
  SolverParams params;
  fill_default_params(&params);

  memset(header, 0, sizeof(header));
  while ((c = getopt(argc, argv, "scd:h:m:n:r:U:Z:z:")) != -1) {
    switch (c) {
      case 's':
        print_log("SYNOPSIS\n  cuda%d [-s] [-c] [-d device] [-h hexheader] [-m trims] [-n nonce] [-r range] [-U seedblocks] [-Z recoverblocks] [-z recoverthreads]\n", EDGEBITS);
        print_log("DEFAULTS\n  cuda%d -d %d -h \"\" -m %d -n %d -r %d -U %d -Z %d -z %d\n", EDGEBITS, device, tp.ntrims, nonce, range, tp.seed.blocks, tp.recover.blocks, tp.recover.tpb);
        exit(0);
      case 'c':
        params.cpuload = false;
        break;
      case 'd':
        device = params.device = atoi(optarg);
        break;
      case 'h':
        len = strlen(optarg)/2;
        assert(len <= sizeof(header));
        for (u32 i=0; i<len; i++)
          sscanf(optarg+2*i, "%2hhx", header+i); // hh specifies storage of a single byte
        break;
      case 'm':
        params.ntrims = atoi(optarg);
        break;
      case 'n':
        nonce = atoi(optarg);
        break;
      case 'r':
        range = atoi(optarg);
        break;
      case 'U':
        params.genablocks = atoi(optarg);
        break;
      case 'Z':
        params.recoverblocks = atoi(optarg);
        break;
      case 'z':
        params.recovertpb = atoi(optarg);
        break;
    }
  }

  assert((params.ntrims & 1) == (PROOFSIZE/2 & 1)); // number of trims must match half cycle length in parity
  int nDevices;
  checkCudaErrors(hipGetDeviceCount(&nDevices));
  assert(device < nDevices);
  hipDeviceProp_t prop;
  checkCudaErrors(hipGetDeviceProperties(&prop, device));
  u64 dbytes = prop.totalGlobalMem;
  int dunit;
  for (dunit=0; dbytes >= 102400; dbytes>>=10,dunit++) ;
  print_log("%s with %d%cB @ %d bits x %dMHz\n", prop.name, (u32)dbytes, " KMGT"[dunit], prop.memoryBusWidth, prop.memoryClockRate/1000);

  print_log("Looking for %d-cycle on cuckatoo%d(\"%s\",%d", PROOFSIZE, EDGEBITS, header, nonce);
  if (range > 1)
    print_log("-%d", nonce+range-1);
  print_log(") with 50%% edges, %d*%d buckets, %d trims, and %d thread blocks.\n", NX, NY, params.ntrims, NX);

  SolverCtx* ctx = create_solver_ctx(&params);

  u64 bytes = ctx->trimmer.globalbytes();
  int unit;
  for (unit=0; bytes >= 102400; bytes>>=10,unit++) ;
  print_log("Using %d%cB of global memory.\n", (u32)bytes, " KMGT"[unit]);

  run_solver(ctx, header, sizeof(header), nonce, range, NULL, NULL);

  return 0;
}
