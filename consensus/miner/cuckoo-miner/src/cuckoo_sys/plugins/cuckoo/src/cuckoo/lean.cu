#include "hip/hip_runtime.h"
// Cuckoo Cycle, a memory-hard proof-of-work
// Copyright (c) 2013-2016 John Tromp

// The edge-trimming memory optimization is due to Dave Andersen
// http://da-data.blogspot.com/2014/03/a-public-review-of-cuckoo-cycle.html

#include <stdint.h>
#include <string.h>
#include "cuckoo.h"
#include "../crypto/siphash.cuh"

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <set>

// algorithm parameters
#ifndef PART_BITS
// #bits used to partition edge set processing to save memory
// a value of 0 does no partitioning and is fastest
// a value of 1 partitions in two, making twice_set the
// same size as shrinkingset at about 33% slowdown
// higher values are not that interesting
#define PART_BITS 0
#endif

#ifndef IDXSHIFT
// we want sizeof(cuckoo_hash) == sizeof(twice_set), so
// CUCKOO_SIZE * sizeof(u64) == TWICE_WORDS * sizeof(u32)
// CUCKOO_SIZE * 2 == TWICE_WORDS
// (NNODES >> IDXSHIFT) * 2 == 2 * ONCE_BITS / 32
// NNODES >> IDXSHIFT == NEDGES >> PART_BITS >> 5
// IDXSHIFT == 1 + PART_BITS + 5
#define IDXSHIFT (PART_BITS + 6)
#endif

#define NODEBITS (EDGEBITS + 1)
#define NNODES (2 * NEDGES)
#define NODEMASK (NNODES-1)

// grow with cube root of size, hardly affected by trimming
#define MAXPATHLEN (8 << (NODEBITS/3))

#define checkCudaErrors(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
  if (code != hipSuccess) {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

// set that starts out full and gets reset by threads on disjoint words
class shrinkingset {
public:
  u32 *bits;
  __device__ void reset(word_t n) {
    bits[n/32] |= 1 << (n%32);
  }
  __device__ bool test(word_t n) const {
    return !((bits[n/32] >> (n%32)) & 1);
  }
  __device__ u32 block(word_t n) const {
    return ~bits[n/32];
  }
};

#define PART_MASK ((1 << PART_BITS) - 1)
#define ONCE_BITS (NEDGES >> PART_BITS)
#define TWICE_WORDS ((2 * ONCE_BITS) / 32)

class twice_set {
public:
  u32 *bits;
  __device__ void reset() {
    memset(bits, 0, TWICE_WORDS * sizeof(u32));
  }
  __device__ void set(word_t u) {
    word_t idx = u/16;
    u32 bit = 1 << (2 * (u%16));
    u32 old = atomicOr(&bits[idx], bit);
    u32 bit2 = bit<<1;
    if ((old & (bit2|bit)) == bit) atomicOr(&bits[idx], bit2);
  }
  __device__ u32 test(word_t u) const {
    return (bits[u/16] >> (2 * (u%16))) & 2;
  }
};

#define CUCKOO_SIZE (NNODES >> IDXSHIFT)
#define CUCKOO_MASK (CUCKOO_SIZE - 1)
// number of (least significant) key bits that survives leftshift by NODEBITS
#define KEYBITS (64-NODEBITS)
#define KEYMASK ((1L << KEYBITS) - 1)
#define MAXDRIFT (1L << (KEYBITS - IDXSHIFT))

class cuckoo_hash {
public:
  u64 *cuckoo;

  cuckoo_hash() {
    cuckoo = (u64 *)calloc(CUCKOO_SIZE, sizeof(u64));
    assert(cuckoo != 0);
  }
  ~cuckoo_hash() {
    free(cuckoo);
  }
  void set(word_t u, word_t v) {
    u64 niew = (u64)u << NODEBITS | v;
    for (word_t ui = u >> IDXSHIFT; ; ui = (ui+1) & CUCKOO_MASK) {
#ifdef ATOMIC
      u64 old = 0;
      if (cuckoo[ui].compare_exchange_strong(old, niew, std::memory_order_relaxed))
        return;
      if ((old >> NODEBITS) == (u & KEYMASK)) {
        cuckoo[ui].store(niew, std::memory_order_relaxed);
#else
      u64 old = cuckoo[ui];
      if (old == 0 || (old >> NODEBITS) == (u & KEYMASK)) {
        cuckoo[ui] = niew;
#endif
        return;
      }
    }
  }
  word_t operator[](word_t u) const {
    for (word_t ui = u >> IDXSHIFT; ; ui = (ui+1) & CUCKOO_MASK) {
#ifdef ATOMIC
      u64 cu = cuckoo[ui].load(std::memory_order_relaxed);
#else
      u64 cu = cuckoo[ui];
#endif
      if (!cu)
        return 0;
      if ((cu >> NODEBITS) == (u & KEYMASK)) {
        assert(((ui - (u >> IDXSHIFT)) & CUCKOO_MASK) < MAXDRIFT);
        return (word_t)(cu & NODEMASK);
      }
    }
  }
};

// arbitrary length of header hashed into siphash key
#define HEADERLEN 80

class cuckoo_ctx {
public:
  siphash_keys sip_keys;
  shrinkingset alive;
  twice_set nonleaf;
  int nthreads;

  cuckoo_ctx(const u32 n_threads) {
    nthreads = n_threads;
  }
  void setheadernonce(char* headernonce, const u32 nonce) {
    ((u32 *)headernonce)[HEADERLEN/sizeof(u32)-1] = htole32(nonce); // place nonce at end
    setheader(headernonce, HEADERLEN, &sip_keys);
  }
};

__global__ void count_node_deg(cuckoo_ctx *ctx, u32 uorv, u32 part) {
  shrinkingset &alive = ctx->alive;
  twice_set &nonleaf = ctx->nonleaf;
  siphash_keys sip_keys = ctx->sip_keys; // local copy sip context; 2.5% speed gain
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  for (word_t block = id*32; block < NEDGES; block += ctx->nthreads*32) {
    u32 alive32 = alive.block(block);
    for (word_t nonce = block-1; alive32; ) { // -1 compensates for 1-based ffs
      u32 ffs = __ffs(alive32);
      nonce += ffs; alive32 >>= ffs;
      word_t u = dipnode(sip_keys, nonce, uorv);
      if ((u & PART_MASK) == part) {
        nonleaf.set(u >> PART_BITS);
      }
    }
  }
}

__global__ void kill_leaf_edges(cuckoo_ctx *ctx, u32 uorv, u32 part) {
  shrinkingset &alive = ctx->alive;
  twice_set &nonleaf = ctx->nonleaf;
  siphash_keys sip_keys = ctx->sip_keys;
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  for (word_t block = id*32; block < NEDGES; block += ctx->nthreads*32) {
    u32 alive32 = alive.block(block);
    for (word_t nonce = block-1; alive32; ) { // -1 compensates for 1-based ffs
      u32 ffs = __ffs(alive32);
      nonce += ffs; alive32 >>= ffs;
      word_t u = dipnode(sip_keys, nonce, uorv);
      if ((u & PART_MASK) == part) {
        if (!nonleaf.test(u >> PART_BITS)) {
          alive.reset(nonce);
        }
      }
    }
  }
}

u32 path(cuckoo_hash &cuckoo, word_t u, word_t *us) {
  u32 nu;
  for (nu = 0; u; u = cuckoo[u]) {
    if (nu >= MAXPATHLEN) {
      while (nu-- && us[nu] != u) ;
      if (nu == ~0)
        printf("maximum path length exceeded\n");
      else printf("illegal % 4d-cycle\n", MAXPATHLEN-nu);
      exit(0);
    }
    us[nu++] = u;
  }
  return nu-1;
}

typedef std::pair<word_t,word_t> edge;

#include <unistd.h>

int main(int argc, char **argv) {
  int nthreads = 16384;
  int trims   = 32;
  int tpb = 0;
  int nonce = 0;
  int range = 1;
  const char *header = "";
  int c;
  while ((c = getopt (argc, argv, "h:n:m:r:t:p:")) != -1) {
    switch (c) {
      case 'h':
        header = optarg;
        break;
      case 'n':
        nonce = atoi(optarg);
        break;
      case 'm':
        trims = atoi(optarg);
        break;
      case 't':
        nthreads = atoi(optarg);
        break;
      case 'p':
        tpb = atoi(optarg);
        break;
      case 'r':
        range = atoi(optarg);
        break;
    }
  }
  if (!tpb) // if not set, then default threads per block to roughly square root of threads
    for (tpb = 1; tpb*tpb < nthreads; tpb *= 2) ;

  printf("Looking for %d-cycle on cuckoo%d(\"%s\",%d", PROOFSIZE, NODEBITS, header, nonce);
  if (range > 1)
    printf("-%d", nonce+range-1);
  printf(") with 50%% edges, %d trims, %d threads %d per block\n", trims, nthreads, tpb);

  cuckoo_ctx ctx(nthreads);

  char headernonce[HEADERLEN];
  u32 hdrlen = strlen(header);
  memcpy(headernonce, header, hdrlen);
  memset(headernonce+hdrlen, 0, sizeof(headernonce)-hdrlen);

  u64 edgeBytes = NEDGES/8, nodeBytes = TWICE_WORDS*sizeof(u32);
  checkCudaErrors(hipMalloc((void**)&ctx.alive.bits, edgeBytes));
  checkCudaErrors(hipMalloc((void**)&ctx.nonleaf.bits, nodeBytes));

  int edgeUnit=0, nodeUnit=0;
  u64 eb = edgeBytes, nb = nodeBytes;
  for (; eb >= 1024; eb>>=10) edgeUnit++;
  for (; nb >= 1024; nb>>=10) nodeUnit++;
  printf("Using %d%cB edge and %d%cB node memory.\n",
     (int)eb, " KMGT"[edgeUnit], (int)nb, " KMGT"[nodeUnit]);

  cuckoo_ctx *device_ctx;
  checkCudaErrors(hipMalloc((void**)&device_ctx, sizeof(cuckoo_ctx)));

  hipEvent_t start, stop;
  checkCudaErrors(hipEventCreate(&start)); checkCudaErrors(hipEventCreate(&stop));
  for (int r = 0; r < range; r++) {
    hipEventRecord(start, NULL);
    checkCudaErrors(hipMemset(ctx.alive.bits, 0, edgeBytes));
    ctx.setheadernonce(headernonce, nonce + r);
    hipMemcpy(device_ctx, &ctx, sizeof(cuckoo_ctx), hipMemcpyHostToDevice);
    for (u32 round=0; round < trims; round++) {
      for (u32 uorv = 0; uorv < 2; uorv++) {
        for (u32 part = 0; part <= PART_MASK; part++) {
          checkCudaErrors(hipMemset(ctx.nonleaf.bits, 0, nodeBytes));
          count_node_deg<<<nthreads/tpb,tpb >>>(device_ctx, uorv, part);
          kill_leaf_edges<<<nthreads/tpb,tpb >>>(device_ctx, uorv, part);
        }
      }
    }
  
    u64 *bits;
    bits = (u64 *)calloc(NEDGES/64, sizeof(u64));
    assert(bits != 0);
    hipMemcpy(bits, ctx.alive.bits, (NEDGES/64) * sizeof(u64), hipMemcpyDeviceToHost);

    checkCudaErrors(hipDeviceSynchronize()); hipEventRecord(stop, NULL);
    float duration;
    hipEventSynchronize(stop); hipEventElapsedTime(&duration, start, stop);
    u32 cnt = 0;
    for (int i = 0; i < NEDGES/64; i++)
      cnt += __builtin_popcountll(~bits[i]);
    u32 load = (u32)(100L * cnt / CUCKOO_SIZE);
    printf("nonce %d: %d trims completed in %.3f seconds final load %d%%\n",
            nonce+r, trims, duration / 1000.0f, load);
  
    if (load >= 90) {
      printf("overloaded! exiting...");
      exit(0);
    }
  
    cuckoo_hash &cuckoo = *(new cuckoo_hash());
    word_t us[MAXPATHLEN], vs[MAXPATHLEN];
    for (word_t block = 0; block < NEDGES; block += 64) {
      u64 alive64 = ~bits[block/64];
      for (word_t nonce = block-1; alive64; ) { // -1 compensates for 1-based ffs
        u32 ffs = __builtin_ffsll(alive64);
        nonce += ffs; alive64 >>= ffs;
        word_t u0=sipnode_(&ctx.sip_keys, nonce, 0), v0=sipnode_(&ctx.sip_keys, nonce, 1);
        if (u0) {
          u32 nu = path(cuckoo, u0, us), nv = path(cuckoo, v0, vs);
          if (us[nu] == vs[nv]) {
            u32 min = nu < nv ? nu : nv;
            for (nu -= min, nv -= min; us[nu] != vs[nv]; nu++, nv++) ;
            u32 len = nu + nv + 1;
            printf("%4d-cycle found at %d:%d%%\n", len, 0, (u32)(nonce*100L/NEDGES));
            if (len == PROOFSIZE) {
              printf("Solution");
              std::set<edge> cycle;
              u32 n = 0;
              cycle.insert(edge(*us, *vs));
              while (nu--)
                cycle.insert(edge(us[(nu+1)&~1], us[nu|1])); // u's in even position; v's in odd
              while (nv--)
                cycle.insert(edge(vs[nv|1], vs[(nv+1)&~1])); // u's in odd position; v's in even
              for (word_t blk = 0; blk < NEDGES; blk += 64) {
                u64 alv64 = ~bits[blk/64];
                for (word_t nce = blk-1; alv64; ) { // -1 compensates for 1-based ffs
                  u32 ffs = __builtin_ffsll(alv64);
                  nce += ffs; alv64 >>= ffs;
                  edge e(sipnode_(&ctx.sip_keys, nce, 0), sipnode_(&ctx.sip_keys, nce, 1));
                  if (cycle.find(e) != cycle.end()) {
                    printf(" %jx", (uintmax_t)nce);
                    if (PROOFSIZE > 2)
                      cycle.erase(e);
                    n++;
                  }
                  if (ffs & 64) break; // can't shift by 64
                }
              }
              assert(n==PROOFSIZE);
              printf("\n");
            }
          } else if (nu < nv) {
            while (nu--)
              cuckoo.set(us[nu+1], us[nu]);
            cuckoo.set(u0, v0);
          } else {
            while (nv--)
              cuckoo.set(vs[nv+1], vs[nv]);
            cuckoo.set(v0, u0);
          }
        }
        if (ffs & 64) break; // can't shift by 64
      }
    }
  }
  checkCudaErrors(hipEventDestroy(start)); checkCudaErrors(hipEventDestroy(stop));
  checkCudaErrors(hipFree(ctx.alive.bits));
  checkCudaErrors(hipFree(ctx.nonleaf.bits));
  return 0;
}
