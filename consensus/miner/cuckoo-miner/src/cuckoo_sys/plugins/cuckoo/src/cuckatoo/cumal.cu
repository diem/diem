#include <stdio.h>
#include <inttypes.h>
#include <assert.h>

int main(int argc, char **argv) {
  size_t bufferMB;
  void *buffer;
  int device = argc > 1 ? atoi(argv[argc-1]) : 1;
  int nDevices;
  hipGetDeviceCount(&nDevices);
  assert(device < nDevices);
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, device);
  uint64_t dbytes = prop.totalGlobalMem;
  int availMB = dbytes >> 20;
  printf("%s with %d MB @ %d bits x %dMHz\n", prop.name, availMB, prop.memoryBusWidth, prop.memoryClockRate/1000);

  hipSetDevice(device);
  for (bufferMB = availMB; ; bufferMB -= 1) {
    int ret = hipMalloc((void**)&buffer, bufferMB << 20);
    if (ret) printf("hipMalloc(%d MB) returned %d\n", bufferMB, ret);
    else break;
  }
  printf("hipMalloc(%d MB) succeeded %d\n", bufferMB);
  hipFree(buffer);

  return 0;
}
