#include "hip/hip_runtime.h"
// Cuckaroo Cycle, a memory-hard proof-of-work by John Tromp
// Copyright (c) 2018-2019 Jiri Vadura (photon) and John Tromp
// This software is covered by the FAIR MINING license

#include <stdio.h>
#include <string.h>
#include <vector>
#include <assert.h>
#include "cuckaroo.hpp"
#include "graph.hpp"
#include "../crypto/siphash.cuh"
#include "../crypto/blake2.h"

typedef uint8_t u8;
typedef uint16_t u16;
typedef uint64_t u64; // save some typing

#ifndef MAXSOLS
#define MAXSOLS 4
#endif

#ifndef IDXSHIFT
// number of bits of compression of surviving edge endpoints
// reduces space used in cycle finding, but too high a value
// results in NODE OVERFLOW warnings and fake cycles
#define IDXSHIFT 12
#endif

const u32 MAXEDGES = NEDGES >> IDXSHIFT;

#ifndef XBITS
#define XBITS 6
#endif

const u32 NX        = 1 << XBITS;
const u32 NX2       = NX * NX;
const u32 XMASK     = NX - 1;
const u32 YBITS     = XBITS;
const u32 NY        = 1 << YBITS;
const u32 YZBITS    = EDGEBITS - XBITS;
const u32 ZBITS     = YZBITS - YBITS;
const u32 NZ        = 1 << ZBITS;
const u32 ZMASK     = NZ - 1;

#ifndef NEPS_A
#define NEPS_A 133
#endif
#ifndef NEPS_B
#define NEPS_B 88
#endif
#define NEPS 128

const u32 EDGES_A = NZ * NEPS_A / NEPS;
const u32 EDGES_B = NZ * NEPS_B / NEPS;

const u32 ROW_EDGES_A = EDGES_A * NY;
const u32 ROW_EDGES_B = EDGES_B * NY;

// Number of Parts of BufferB, all but one of which will overlap BufferA
#ifndef NB
#define NB 2
#endif

#ifndef NA
#define NA  ((NB * NEPS_A + NEPS_B-1) / NEPS_B)
#endif

__constant__ uint2 recoveredges[PROOFSIZE];
__constant__ uint2 e0 = {0,0};

__device__ u64 dipblock(const siphash_keys &keys, const word_t edge, u64 *buf) {
  diphash_state<> shs(keys);
  word_t edge0 = edge & ~EDGE_BLOCK_MASK;
  u32 i;
  for (i=0; i < EDGE_BLOCK_MASK; i++) {
    shs.hash24(edge0 + i);
    buf[i] = shs.xor_lanes();
  }
  shs.hash24(edge0 + i);
  buf[i] = 0;
  return shs.xor_lanes();
}

__device__ u32 endpoint(uint2 nodes, int uorv) {
  return uorv ? nodes.y : nodes.x;
}

#ifndef FLUSHA // should perhaps be in trimparams and passed as template parameter
#define FLUSHA 16
#endif

template<int maxOut>
__global__ void SeedA(const siphash_keys &sipkeys, ulonglong4 * __restrict__ buffer, u32 * __restrict__ indexes) {
  const int group = blockIdx.x;
  const int dim = blockDim.x;
  const int lid = threadIdx.x;
  const int gid = group * dim + lid;
  const int nthreads = gridDim.x * dim;
  const int FLUSHA2 = 2*FLUSHA;

  __shared__ uint2 tmp[NX][FLUSHA2]; // needs to be ulonglong4 aligned
  const int TMPPERLL4 = sizeof(ulonglong4) / sizeof(uint2);
  __shared__ int counters[NX];
  u64 buf[EDGE_BLOCK_SIZE];

  for (int row = lid; row < NX; row += dim)
    counters[row] = 0;
  __syncthreads();

  const int col = group % NX;
  const int loops = NEDGES / nthreads; // assuming THREADS_HAVE_EDGES checked
  for (int blk = 0; blk < loops; blk += EDGE_BLOCK_SIZE) {
    u32 nonce0 = gid * loops + blk;
    const u64 last = dipblock(sipkeys, nonce0, buf);
    for (u32 e = 0; e < EDGE_BLOCK_SIZE; e++) {
      u64 edge = buf[e] ^ last;
      u32 node0 = edge & EDGEMASK;
      u32 node1 = (edge >> 32) & EDGEMASK;
      int row = node0 >> YZBITS;
      int counter = min((int)atomicAdd(counters + row, 1), (int)(FLUSHA2-1)); // assuming ROWS_LIMIT_LOSSES checked
      tmp[row][counter] = make_uint2(node0, node1);
      __syncthreads();
      if (counter == FLUSHA-1) {
        int localIdx = min(FLUSHA2, counters[row]);
        int newCount = localIdx % FLUSHA;
        int nflush = localIdx - newCount;
        u32 grp = row * NX + col;
        int cnt = min((int)atomicAdd(indexes + grp, nflush), (int)(maxOut - nflush));
        for (int i = 0; i < nflush; i += TMPPERLL4)
          buffer[((u64)grp * maxOut + cnt + i) / TMPPERLL4] = *(ulonglong4 *)(&tmp[row][i]);
        for (int t = 0; t < newCount; t++) {
          tmp[row][t] = tmp[row][t + nflush];
        }
        counters[row] = newCount;
      }
      __syncthreads();
    }
  }
  uint2 zero = make_uint2(0, 0);
  for (int row = lid; row < NX; row += dim) {
    int localIdx = min(FLUSHA2, counters[row]);
    u32 grp = row * NX + col;
    for (int j = localIdx; j % TMPPERLL4; j++)
      tmp[row][j] = zero;
    for (int i = 0; i < localIdx; i += TMPPERLL4) {
      int cnt = min((int)atomicAdd(indexes + grp, TMPPERLL4), (int)(maxOut - TMPPERLL4));
      buffer[((u64)grp * maxOut + cnt) / TMPPERLL4] = *(ulonglong4 *)(&tmp[row][i]);
    }
  }
}

template <typename Edge> __device__ bool null(Edge e);

__device__ bool null(u32 nonce) {
  return nonce == 0;
}

__device__ bool null(uint2 nodes) {
  return nodes.x == 0 && nodes.y == 0;
}

#ifndef FLUSHB
#define FLUSHB 8
#endif

template<int maxOut>
__global__ void SeedB(const uint2 * __restrict__ source, ulonglong4 * __restrict__ destination, const u32 * __restrict__ srcIdx, u32 * __restrict__ dstIdx) {
  const int group = blockIdx.x;
  const int dim = blockDim.x;
  const int lid = threadIdx.x;
  const int FLUSHB2 = 2 * FLUSHB;

  __shared__ uint2 tmp[NX][FLUSHB2];
  const int TMPPERLL4 = sizeof(ulonglong4) / sizeof(uint2);
  __shared__ int counters[NX];

  for (int col = lid; col < NX; col += dim)
    counters[col] = 0;
  __syncthreads();
  const int row = group / NX;
  const int bucketEdges = min((int)srcIdx[group], (int)maxOut);
  const int loops = (bucketEdges + dim-1) / dim;
  for (int loop = 0; loop < loops; loop++) {
    int col;
    int counter = 0;
    const int edgeIndex = loop * dim + lid;
    if (edgeIndex < bucketEdges) {
      const int index = group * maxOut + edgeIndex;
      uint2 edge = __ldg(&source[index]);
      if (!null(edge)) {
        u32 node1 = edge.x;
        col = (node1 >> ZBITS) & XMASK;
        counter = min((int)atomicAdd(counters + col, 1), (int)(FLUSHB2-1)); // assuming COLS_LIMIT_LOSSES checked
        tmp[col][counter] = edge;
        }
    }
    __syncthreads();
    if (counter == FLUSHB-1) {
      int localIdx = min(FLUSHB2, counters[col]);
      int newCount = localIdx % FLUSHB;
      int nflush = localIdx - newCount;
      u32 grp = row * NX + col;
#ifdef SYNCBUG
      if (grp==0x2d6) printf("group %x size %d lid %d nflush %d\n", group, bucketEdges, lid, nflush);
#endif
      int cnt = min((int)atomicAdd(dstIdx + grp, nflush), (int)(maxOut - nflush));
      for (int i = 0; i < nflush; i += TMPPERLL4)
        destination[((u64)grp * maxOut + cnt + i) / TMPPERLL4] = *(ulonglong4 *)(&tmp[col][i]);
      for (int t = 0; t < newCount; t++) {
        tmp[col][t] = tmp[col][t + nflush];
      }
      counters[col] = newCount;
    }
    __syncthreads(); 
  }
  uint2 zero = make_uint2(0, 0);
  for (int col = lid; col < NX; col += dim) {
    int localIdx = min(FLUSHB2, counters[col]);
    u32 grp = row * NX + col;
#ifdef SYNCBUG
    if (group==0x2f2 && grp==0x2d6) printf("group %x size %d lid %d localIdx %d\n", group, bucketEdges, lid, localIdx);
#endif
    for (int j = localIdx; j % TMPPERLL4; j++)
      tmp[col][j] = zero;
    for (int i = 0; i < localIdx; i += TMPPERLL4) {
      int cnt = min((int)atomicAdd(dstIdx + grp, TMPPERLL4), (int)(maxOut - TMPPERLL4));
      destination[((u64)grp * maxOut + cnt) / TMPPERLL4] = *(ulonglong4 *)(&tmp[col][i]);
    }
  }
}

__device__ __forceinline__  void Increase2bCounter(u32 *ecounters, const int bucket) {
  int word = bucket >> 5;
  unsigned char bit = bucket & 0x1F;
  u32 mask = 1 << bit;

  u32 old = atomicOr(ecounters + word, mask) & mask;
  if (old)
    atomicOr(ecounters + word + NZ/32, mask);
}

__device__ __forceinline__  bool Read2bCounter(u32 *ecounters, const int bucket) {
  int word = bucket >> 5;
  unsigned char bit = bucket & 0x1F;

  return (ecounters[word + NZ/32] >> bit) & 1;
}

template<int NP, int maxIn, int maxOut>
__global__ void Round(const int round, const uint2 * __restrict__ src, uint2 * __restrict__ dst, const u32 * __restrict__ srcIdx, u32 * __restrict__ dstIdx) {
  const int group = blockIdx.x;
  const int dim = blockDim.x;
  const int lid = threadIdx.x;
  const int COUNTERWORDS = NZ / 16; // 16 2-bit counters per 32-bit word

  __shared__ u32 ecounters[COUNTERWORDS];

  for (int i = lid; i < COUNTERWORDS; i += dim)
    ecounters[i] = 0;
  __syncthreads();

  for (int i = 0; i < NP; i++, src += NX2 * maxIn, srcIdx += NX2) {
    const int edgesInBucket = min(srcIdx[group], maxIn);
    // if (!group && !lid) printf("round %d size  %d\n", round, edgesInBucket);
    const int loops = (edgesInBucket + dim-1) / dim;

    for (int loop = 0; loop < loops; loop++) {
      const int lindex = loop * dim + lid;
      if (lindex < edgesInBucket) {
        const int index = maxIn * group + lindex;
        uint2 edge = __ldg(&src[index]);
        if (null(edge)) continue;
        u32 node = endpoint(edge, round&1);
        Increase2bCounter(ecounters, node & ZMASK);
      }
    }
  }

  __syncthreads();

  src -= NP * NX2 * maxIn; srcIdx -= NP * NX2;
  for (int i = 0; i < NP; i++, src += NX2 * maxIn, srcIdx += NX2) {
    const int edgesInBucket = min(srcIdx[group], maxIn);
    const int loops = (edgesInBucket + dim-1) / dim;
    for (int loop = 0; loop < loops; loop++) {
      const int lindex = loop * dim + lid;
      if (lindex < edgesInBucket) {
        const int index = maxIn * group + lindex;
        uint2 edge = __ldg(&src[index]);
        if (null(edge)) continue;
        u32 node0 = endpoint(edge, round&1);
        if (Read2bCounter(ecounters, node0 & ZMASK)) {
          u32 node1 = endpoint(edge, (round&1)^1);
          const int bucket = node1 >> ZBITS;
          const int bktIdx = min(atomicAdd(dstIdx + bucket, 1), maxOut - 1);
          dst[bucket * maxOut + bktIdx] = (round&1) ? make_uint2(node1, node0) : make_uint2(node0, node1);
        }
      }
    }
  }
}

template<int maxIn>
__global__ void Tail(const uint2 *source, uint2 *destination, const u32 *srcIdx, u32 *dstIdx) {
  const int lid = threadIdx.x;
  const int group = blockIdx.x;
  const int dim = blockDim.x;
  int myEdges = srcIdx[group];
  __shared__ int destIdx;

  if (lid == 0)
    destIdx = atomicAdd(dstIdx, myEdges);
  __syncthreads();
  for (int i = lid; i < myEdges; i += dim)
    destination[destIdx + i] = source[group * maxIn + i];
}

#define checkCudaErrors_V(ans) ({if (gpuAssert((ans), __FILE__, __LINE__) != hipSuccess) return;})
#define checkCudaErrors_N(ans) ({if (gpuAssert((ans), __FILE__, __LINE__) != hipSuccess) return NULL;})
#define checkCudaErrors(ans) ({int retval = gpuAssert((ans), __FILE__, __LINE__); if (retval != hipSuccess) return retval;})

inline int gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
  int device_id;
  hipGetDevice(&device_id);
  if (code != hipSuccess) {
    snprintf(LAST_ERROR_REASON, MAX_NAME_LEN, "Device %d GPUassert: %s %s %d", device_id, hipGetErrorString(code), file, line);
    hipDeviceReset();
    if (abort) return code;
  }
  return code;
}

__global__ void Recovery(const siphash_keys &sipkeys, ulonglong4 *buffer, int *indexes) {
  const int gid = blockDim.x * blockIdx.x + threadIdx.x;
  const int lid = threadIdx.x;
  const int nthreads = blockDim.x * gridDim.x;
  const int loops = NEDGES / nthreads;
  __shared__ u32 nonces[PROOFSIZE];
  u64 buf[EDGE_BLOCK_SIZE];

  if (lid < PROOFSIZE) nonces[lid] = 0;
  __syncthreads();
  for (int blk = 0; blk < loops; blk += EDGE_BLOCK_SIZE) {
    u32 nonce0 = gid * loops + blk;
    const u64 last = dipblock(sipkeys, nonce0, buf);
    for (int i = 0; i < EDGE_BLOCK_SIZE; i++) {
      u64 edge = buf[i] ^ last;
      u32 u = edge & EDGEMASK;
      u32 v = (edge >> 32) & EDGEMASK;
      for (int p = 0; p < PROOFSIZE; p++) { //YO
        if (recoveredges[p].x == u && recoveredges[p].y == v) {
          nonces[p] = nonce0 + i;
        }
      }
    }
  }
  __syncthreads();
  if (lid < PROOFSIZE) {
    if (nonces[lid] > 0)
      indexes[lid] = nonces[lid];
  }
}

struct blockstpb {
  u16 blocks;
  u16 tpb;
};

struct trimparams {
  u16 ntrims;
  blockstpb genA;
  blockstpb genB;
  blockstpb trim;
  blockstpb tail;
  blockstpb recover;

  trimparams() {
    ntrims              =  176;
    genA.blocks         = 4096;
    genA.tpb            =  256;
    genB.blocks         =  NX2;
    genB.tpb            =  128;
    trim.blocks         =  NX2;
    trim.tpb            =  512;
    tail.blocks         =  NX2;
    tail.tpb            = 1024;
    recover.blocks      = 1024;
    recover.tpb         = 1024;
  }
};

typedef u32 proof[PROOFSIZE];

// maintains set of trimmable edges
struct edgetrimmer {
  trimparams tp;
  edgetrimmer *dt;
  size_t sizeA, sizeB;
  const size_t indexesSize = NX * NY * sizeof(u32);
  u8 *bufferA;
  u8 *bufferB;
  u8 *bufferAB;
  u32 *indexesE[1+NB];
  u32 nedges;
  u32 *uvnodes;
  siphash_keys sipkeys, *dipkeys;
  bool abort;
  bool initsuccess = false;

  edgetrimmer(const trimparams _tp) : tp(_tp) {
    checkCudaErrors_V(hipMalloc((void**)&dt, sizeof(edgetrimmer)));
    checkCudaErrors_V(hipMalloc((void**)&uvnodes, PROOFSIZE * 2 * sizeof(u32)));
    checkCudaErrors_V(hipMalloc((void**)&dipkeys, sizeof(siphash_keys)));
    for (int i = 0; i < 1+NB; i++) {
      checkCudaErrors_V(hipMalloc((void**)&indexesE[i], indexesSize));
    }
    sizeA = ROW_EDGES_A * NX * sizeof(uint2);
    sizeB = ROW_EDGES_B * NX * sizeof(uint2);
    const size_t bufferSize = sizeA + sizeB / NB;
    assert(bufferSize >= sizeB + sizeB / NB / 2); // ensure enough space for Round 1
    checkCudaErrors_V(hipMalloc((void**)&bufferA, bufferSize));
    bufferAB = bufferA + sizeB / NB;
    bufferB  = bufferA + bufferSize - sizeB;
    assert(bufferA + sizeA == bufferB + sizeB * (NB-1) / NB); // ensure alignment of overlap
    hipMemcpy(dt, this, sizeof(edgetrimmer), hipMemcpyHostToDevice);
    initsuccess = true;
  }
  u64 globalbytes() const {
    return (sizeA+sizeB/NB) + (1+NB) * indexesSize + sizeof(siphash_keys) + PROOFSIZE * 2 * sizeof(u32) + sizeof(edgetrimmer);
  }
  ~edgetrimmer() {
    checkCudaErrors_V(hipFree(bufferA));
    for (int i = 0; i < 1+NB; i++) {
      checkCudaErrors_V(hipFree(indexesE[i]));
    }
    checkCudaErrors_V(hipFree(dipkeys));
    checkCudaErrors_V(hipFree(uvnodes));
    checkCudaErrors_V(hipFree(dt));
    hipDeviceReset();
  }
  u32 trim() {
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start)); checkCudaErrors(hipEventCreate(&stop));
    hipMemcpy(dipkeys, &sipkeys, sizeof(sipkeys), hipMemcpyHostToDevice);
  
    hipDeviceSynchronize();
    float durationA, durationB;
    hipEventRecord(start, NULL);
  
    hipMemset(indexesE[1], 0, indexesSize);

    SeedA<EDGES_A><<<tp.genA.blocks, tp.genA.tpb>>>(*dipkeys, (ulonglong4*)bufferAB, indexesE[1]);
  
    checkCudaErrors(hipDeviceSynchronize()); hipEventRecord(stop, NULL);
    hipEventSynchronize(stop); hipEventElapsedTime(&durationA, start, stop);
    if (abort) return false;
    hipEventRecord(start, NULL);
  
    hipMemset(indexesE[0], 0, indexesSize);

    u32 qA = sizeA/NA;
    u32 qE = NX2 / NA;
    for (u32 i = 0; i < NA; i++) {
      SeedB<EDGES_A><<<tp.genB.blocks/NA, tp.genB.tpb>>>((uint2*)(bufferAB+i*qA), (ulonglong4*)(bufferA+i*qA), indexesE[1]+i*qE, indexesE[0]+i*qE);
      if (abort) return false;
    }

    checkCudaErrors(hipDeviceSynchronize()); hipEventRecord(stop, NULL);
    hipEventSynchronize(stop); hipEventElapsedTime(&durationB, start, stop);
    checkCudaErrors(hipEventDestroy(start)); checkCudaErrors(hipEventDestroy(stop));
    print_log("Seeding completed in %.0f + %.0f ms\n", durationA, durationB);
    if (abort) return false;
  
    for (u32 i = 0; i < NB; i++) hipMemset(indexesE[1+i], 0, indexesSize);

    qA = sizeA/NB;
    const size_t qB = sizeB/NB;
    qE = NX2 / NB;
    for (u32 i = NB; i--; ) {
      Round<1, EDGES_A, EDGES_B/NB><<<tp.trim.blocks/NB, tp.trim.tpb>>>(0, (uint2*)(bufferA+i*qA), (uint2*)(bufferB+i*qB), indexesE[0]+i*qE, indexesE[1+i]); // to .632
      if (abort) return false;
    }

    hipMemset(indexesE[0], 0, indexesSize);

    Round<NB, EDGES_B/NB, EDGES_B/2><<<tp.trim.blocks, tp.trim.tpb>>>(1, (const uint2 *)bufferB, (uint2 *)bufferA, indexesE[1], indexesE[0]); // to .296
    if (abort) return false;

    hipMemset(indexesE[1], 0, indexesSize);

    Round<1, EDGES_B/2, EDGES_A/4><<<tp.trim.blocks, tp.trim.tpb>>>(2, (const uint2 *)bufferA, (uint2 *)bufferB, indexesE[0], indexesE[1]); // to .176
    if (abort) return false;

    hipMemset(indexesE[0], 0, indexesSize);

    Round<1, EDGES_A/4, EDGES_B/4><<<tp.trim.blocks, tp.trim.tpb>>>(3, (const uint2 *)bufferB, (uint2 *)bufferA, indexesE[1], indexesE[0]); // to .117
    if (abort) return false;
  
    hipDeviceSynchronize();
  
    for (int round = 4; round < tp.ntrims; round += 2) {
      hipMemset(indexesE[1], 0, indexesSize);
      Round<1, EDGES_B/4, EDGES_B/4><<<tp.trim.blocks, tp.trim.tpb>>>(round, (const uint2 *)bufferA, (uint2 *)bufferB, indexesE[0], indexesE[1]);
      if (abort) return false;
      hipMemset(indexesE[0], 0, indexesSize);
      Round<1, EDGES_B/4, EDGES_B/4><<<tp.trim.blocks, tp.trim.tpb>>>(round+1, (const uint2 *)bufferB, (uint2 *)bufferA, indexesE[1], indexesE[0]);
      if (abort) return false;
    }
    
    hipMemset(indexesE[1], 0, indexesSize);
    hipDeviceSynchronize();
  
    Tail<EDGES_B/4><<<tp.tail.blocks, tp.tail.tpb>>>((const uint2 *)bufferA, (uint2 *)bufferB, indexesE[0], indexesE[1]);
    hipMemcpy(&nedges, indexesE[1], sizeof(u32), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    return nedges;
  }
};

struct solver_ctx {
  edgetrimmer trimmer;
  bool mutatenonce;
  uint2 *edges;
  graph<word_t> cg;
  uint2 soledges[PROOFSIZE];
  std::vector<u32> sols; // concatenation of all proof's indices

  solver_ctx(const trimparams tp, bool mutate_nonce) : trimmer(tp), cg(MAXEDGES, MAXEDGES, MAXSOLS, IDXSHIFT) {
    edges   = new uint2[MAXEDGES];
    mutatenonce = mutate_nonce;
  }

  void setheadernonce(char * const headernonce, const u32 len, const u32 nonce) {
    if (mutatenonce)
      ((u32 *)headernonce)[len/sizeof(u32)-1] = htole32(nonce); // place nonce at end
    setheader(headernonce, len, &trimmer.sipkeys);
    sols.clear();
  }
  ~solver_ctx() {
    delete[] edges;
  }

  int findcycles(uint2 *edges, u32 nedges) {
    cg.reset();
    for (u32 i = 0; i < nedges; i++)
      cg.add_compress_edge(edges[i].x, edges[i].y);
    for (u32 s = 0 ;s < cg.nsols; s++) {
      // print_log("Solution");
      for (u32 j = 0; j < PROOFSIZE; j++) {
        soledges[j] = edges[cg.sols[s][j]];
        // print_log(" (%x, %x)", soledges[j].x, soledges[j].y);
      }
      // print_log("\n");
      sols.resize(sols.size() + PROOFSIZE);
      hipMemcpyToSymbol(HIP_SYMBOL(recoveredges), soledges, sizeof(soledges));
      hipMemset(trimmer.indexesE[1], 0, trimmer.indexesSize);
      Recovery<<<trimmer.tp.recover.blocks, trimmer.tp.recover.tpb>>>(*trimmer.dipkeys, (ulonglong4*)trimmer.bufferA, (int *)trimmer.indexesE[1]);
      hipMemcpy(&sols[sols.size()-PROOFSIZE], trimmer.indexesE[1], PROOFSIZE * sizeof(u32), hipMemcpyDeviceToHost);
      checkCudaErrors(hipDeviceSynchronize());
      qsort(&sols[sols.size()-PROOFSIZE], PROOFSIZE, sizeof(u32), cg.nonce_cmp);
    }
    return 0;
  }

  int solve() {
    u64 time0, time1;
    u32 timems,timems2;

    trimmer.abort = false;
    time0 = timestamp();
    u32 nedges = trimmer.trim();
    if (!nedges)
      return 0;
    if (nedges > MAXEDGES) {
      print_log("OOPS; losing %d edges beyond MAXEDGES=%d\n", nedges-MAXEDGES, MAXEDGES);
      nedges = MAXEDGES;
    }
    hipMemcpy(edges, trimmer.bufferB, sizeof(uint2[nedges]), hipMemcpyDeviceToHost);
    time1 = timestamp(); timems  = (time1 - time0) / 1000000;
    time0 = timestamp();
    findcycles(edges, nedges);
    time1 = timestamp(); timems2 = (time1 - time0) / 1000000;
    print_log("findcycles edges %d time %d ms total %d ms\n", nedges, timems2, timems+timems2);
    return sols.size() / PROOFSIZE;
  }

  void abort() {
    trimmer.abort = true;
  }
};

#include <unistd.h>

// arbitrary length of header hashed into siphash key
#define HEADERLEN 80

typedef solver_ctx SolverCtx;

CALL_CONVENTION int run_solver(SolverCtx* ctx,
                               char* header,
                               int header_length,
                               u32 nonce,
                               u32 range,
                               SolverSolutions *solutions,
                               SolverStats *stats
                               )
{
  u64 time0, time1;
  u32 timems;
  u32 sumnsols = 0;
  int device_id;
  if (stats != NULL) {
    hipGetDevice(&device_id);
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, stats->device_id);
    stats->device_id = device_id;
    stats->edge_bits = EDGEBITS;
    strncpy(stats->device_name, props.name, MAX_NAME_LEN);
  }

  if (ctx == NULL || !ctx->trimmer.initsuccess){
    print_log("Error initialising trimmer. Aborting.\n");
    print_log("Reason: %s\n", LAST_ERROR_REASON);
    if (stats != NULL) {
       stats->has_errored = true;
       strncpy(stats->error_reason, LAST_ERROR_REASON, MAX_NAME_LEN);
    }
    return 0;
  }

  for (u32 r = 0; r < range; r++) {
    time0 = timestamp();
    ctx->setheadernonce(header, header_length, nonce + r);
    print_log("nonce %d k0 k1 k2 k3 %llx %llx %llx %llx\n", nonce+r, ctx->trimmer.sipkeys.k0, ctx->trimmer.sipkeys.k1, ctx->trimmer.sipkeys.k2, ctx->trimmer.sipkeys.k3);
    u32 nsols = ctx->solve();
    time1 = timestamp();
    timems = (time1 - time0) / 1000000;
    print_log("Time: %d ms\n", timems);
    for (unsigned s = 0; s < nsols; s++) {
      print_log("Solution");
      u32* prf = &ctx->sols[s * PROOFSIZE];
      for (u32 i = 0; i < PROOFSIZE; i++)
        print_log(" %jx", (uintmax_t)prf[i]);
      print_log("\n");
      if (solutions != NULL){
        solutions->edge_bits = EDGEBITS;
        solutions->num_sols++;
        solutions->sols[sumnsols+s].nonce = nonce + r;
        for (u32 i = 0; i < PROOFSIZE; i++) 
          solutions->sols[sumnsols+s].proof[i] = (u64) prf[i];
      }
      int pow_rc = verify(prf, ctx->trimmer.sipkeys);
      if (pow_rc == POW_OK) {
        print_log("Verified with cyclehash ");
        unsigned char cyclehash[32];
        blake2b((void *)cyclehash, sizeof(cyclehash), (const void *)prf, sizeof(proof), 0, 0);
        for (int i=0; i<32; i++)
          print_log("%02x", cyclehash[i]);
        print_log("\n");
      } else {
        print_log("FAILED due to %s\n", errstr[pow_rc]);
      }
    }
    sumnsols += nsols;
    if (stats != NULL) {
      stats->last_start_time = time0;
      stats->last_end_time = time1;
      stats->last_solution_time = time1 - time0;
    }
  }
  print_log("%d total solutions\n", sumnsols);
  return sumnsols > 0;
}

CALL_CONVENTION SolverCtx* create_solver_ctx(SolverParams* params) {
  trimparams tp;
  tp.ntrims = params->ntrims;
  tp.genA.blocks = params->genablocks;
  tp.genA.tpb = params->genatpb;
  tp.genB.tpb = params->genbtpb;
  tp.trim.tpb = params->trimtpb;
  tp.tail.tpb = params->tailtpb;
  tp.recover.blocks = params->recoverblocks;
  tp.recover.tpb = params->recovertpb;

  hipDeviceProp_t prop;
  checkCudaErrors_N(hipGetDeviceProperties(&prop, params->device));

  assert(tp.genA.tpb <= prop.maxThreadsPerBlock);
  assert(tp.genB.tpb <= prop.maxThreadsPerBlock);
  assert(tp.trim.tpb <= prop.maxThreadsPerBlock);
  // assert(tp.tailblocks <= prop.threadDims[0]);
  assert(tp.tail.tpb <= prop.maxThreadsPerBlock);
  assert(tp.recover.tpb <= prop.maxThreadsPerBlock);

  assert(tp.genA.blocks * tp.genA.tpb * EDGE_BLOCK_SIZE <= NEDGES); // check THREADS_HAVE_EDGES
  assert(tp.recover.blocks * tp.recover.tpb * EDGE_BLOCK_SIZE <= NEDGES); // check THREADS_HAVE_EDGES
  assert(tp.genA.tpb / NX <= FLUSHA); // check ROWS_LIMIT_LOSSES
  assert(tp.genB.tpb / NX <= FLUSHB); // check COLS_LIMIT_LOSSES

  hipSetDevice(params->device);
  if (!params->cpuload)
    checkCudaErrors_N(hipSetDeviceFlags(hipDeviceScheduleBlockingSync));

  SolverCtx* ctx = new SolverCtx(tp, params->mutate_nonce);

  return ctx;
}

CALL_CONVENTION void destroy_solver_ctx(SolverCtx* ctx) {
  delete ctx;
}

CALL_CONVENTION void stop_solver(SolverCtx* ctx) {
  ctx->abort();
}

CALL_CONVENTION void fill_default_params(SolverParams* params) {
  trimparams tp;
  params->device = 0;
  params->ntrims = tp.ntrims;
  params->genablocks = min(tp.genA.blocks, NEDGES/EDGE_BLOCK_SIZE/tp.genA.tpb);
  params->genatpb = tp.genA.tpb;
  params->genbtpb = tp.genB.tpb;
  params->trimtpb = tp.trim.tpb;
  params->tailtpb = tp.tail.tpb;
  params->recoverblocks = min(tp.recover.blocks, NEDGES/EDGE_BLOCK_SIZE/tp.recover.tpb);
  params->recovertpb = tp.recover.tpb;
  params->cpuload = false;
}

int main(int argc, char **argv) {
  trimparams tp;
  u32 nonce = 0;
  u32 range = 1;
  u32 device = 0;
  char header[HEADERLEN];
  u32 len;
  int c;

  // set defaults
  SolverParams params;
  fill_default_params(&params);

  memset(header, 0, sizeof(header));
  while ((c = getopt(argc, argv, "scb:d:h:k:m:n:r:U:u:v:w:y:Z:z:")) != -1) {
    switch (c) {
      case 's':
        print_log("SYNOPSIS\n  cuda%d [-s] [-c] [-d device] [-h hexheader] [-m trims] [-n nonce] [-r range] [-U seedAblocks] [-u seedAthreads] [-v seedBthreads] [-w Trimthreads] [-y Tailthreads] [-Z recoverblocks] [-z recoverthreads]\n", EDGEBITS);
        print_log("DEFAULTS\n  cuda%d -d %d -h \"\" -m %d -n %d -r %d -U %d -u %d -v %d -w %d -y %d -Z %d -z %d\n", EDGEBITS, device, tp.ntrims, nonce, range, tp.genA.blocks, tp.genA.tpb, tp.genB.tpb, tp.trim.tpb, tp.tail.tpb, tp.recover.blocks, tp.recover.tpb);
        exit(0);
      case 'c':
        params.cpuload = false;
        break;
      case 'd':
        device = params.device = atoi(optarg);
        break;
      case 'h':
        len = strlen(optarg)/2;
        assert(len <= sizeof(header));
        for (u32 i=0; i<len; i++)
          sscanf(optarg+2*i, "%2hhx", header+i); // hh specifies storage of a single byte
        break;
      case 'n':
        nonce = atoi(optarg);
        break;
      case 'm':
        params.ntrims = atoi(optarg) & -2; // make even as required by solve()
        break;
      case 'r':
        range = atoi(optarg);
        break;
      case 'U':
        params.genablocks = atoi(optarg);
        break;
      case 'u':
        params.genatpb = atoi(optarg);
        break;
      case 'v':
        params.genbtpb = atoi(optarg);
        break;
      case 'w':
        params.trimtpb = atoi(optarg);
        break;
      case 'y':
        params.tailtpb = atoi(optarg);
        break;
      case 'Z':
        params.recoverblocks = atoi(optarg);
        break;
      case 'z':
        params.recovertpb = atoi(optarg);
        break;
    }
  }

  int nDevices;
  checkCudaErrors(hipGetDeviceCount(&nDevices));
  assert(device < nDevices);
  hipDeviceProp_t prop;
  checkCudaErrors(hipGetDeviceProperties(&prop, device));
  u64 dbytes = prop.totalGlobalMem;
  int dunit;
  for (dunit=0; dbytes >= 102040; dbytes>>=10,dunit++) ;
  print_log("%s with %d%cB @ %d bits x %dMHz\n", prop.name, (u32)dbytes, " KMGT"[dunit], prop.memoryBusWidth, prop.memoryClockRate/1000);
  // hipSetDevice(device);

  print_log("Looking for %d-cycle on cuckaroo%d(\"%s\",%d", PROOFSIZE, EDGEBITS, header, nonce);
  if (range > 1)
    print_log("-%d", nonce+range-1);
  print_log(") with 50%% edges, %d*%d buckets, %d trims, and %d thread blocks.\n", NX, NY, params.ntrims, NX);

  SolverCtx* ctx = create_solver_ctx(&params);

  u64 bytes = ctx->trimmer.globalbytes();
  int unit;
  for (unit=0; bytes >= 102400; bytes>>=10,unit++) ;
  print_log("Using %d%cB of global memory.\n", (u32)bytes, " KMGT"[unit]);

  run_solver(ctx, header, sizeof(header), nonce, range, NULL, NULL);

  return 0;
}
