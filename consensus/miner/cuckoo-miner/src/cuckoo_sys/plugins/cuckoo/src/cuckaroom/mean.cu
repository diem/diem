#include "hip/hip_runtime.h"
// Cuckaroom Cycle, a memory-hard proof-of-work by John Tromp
// Copyright (c) 2018-2019 Jiri Vadura (photon) and John Tromp
// This software is covered by the FAIR MINING license

#include <stdio.h>
#include <string.h>
#include <vector>
#include <assert.h>
#include "cuckaroom.hpp"
#include "graph.hpp"
#include "../crypto/blake2.h"

// Number of Parts of BufferB, all but one of which will overlap BufferA
#ifndef NA
#define NA 4
#endif
#define NA2 (NA * NA)

#define NODE1MASK NODEMASK
#include "../crypto/siphash.cuh"

#include "kernel.cuh"

typedef uint8_t u8;
typedef uint16_t u16;

#ifndef IDXSHIFT
// number of bits of compression of surviving edge endpoints
// reduces space used in cycle finding, but too high a value
// results in NODE OVERFLOW warnings and fake cycles
#define IDXSHIFT 12
#endif

const u32 MAXEDGES = NEDGES >> IDXSHIFT;

#ifndef NEPS_A
#define NEPS_A 135
#endif
#ifndef NEPS_B
#define NEPS_B 88
#endif
#ifndef NEPS_C
#define NEPS_C 55
#endif
#define NEPS 128

const u32 EDGES_A = NZ * NEPS_A / NEPS;
const u32 EDGES_B = NZ * NEPS_B / NEPS;
const u32 EDGES_C = NZ * NEPS_C / NEPS;

const u32 ROW_EDGES_A = EDGES_A * NY;
const u32 ROW_EDGES_B = EDGES_B * NY;
const u32 ROW_EDGES_C = EDGES_C * NY;

#define checkCudaErrors_V(ans) ({if (gpuAssert((ans), __FILE__, __LINE__) != hipSuccess) return;})
#define checkCudaErrors_N(ans) ({if (gpuAssert((ans), __FILE__, __LINE__) != hipSuccess) return NULL;})
#define checkCudaErrors(ans) ({int retval = gpuAssert((ans), __FILE__, __LINE__); if (retval != hipSuccess) return retval;})

inline int gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
  int device_id;
  hipGetDevice(&device_id);
  if (code != hipSuccess) {
    snprintf(LAST_ERROR_REASON, MAX_NAME_LEN, "Device %d GPUassert: %s %s %d", device_id, hipGetErrorString(code), file, line);
    hipDeviceReset();
    if (abort) return code;
  }
  return code;
}

struct blockstpb {
  u16 blocks;
  u16 tpb;
};

#ifndef SEED_TPB
#define SEED_TPB 256
#endif
#ifndef TRIM0_TPB
#define TRIM0_TPB 1024
#endif
#ifndef TRIM1_TPB
#define TRIM1_TPB 512
#endif
#ifndef TRIM_TPB
#define TRIM_TPB 512
#endif
#ifndef RELAY_TPB
#define RELAY_TPB 512
#endif
#ifndef TAIL_TPB
#define TAIL_TPB SEED_TPB
#endif

struct trimparams {
  u16 ntrims;
  blockstpb seed;
  blockstpb trim0;
  blockstpb trim1;
  blockstpb trim;
  blockstpb tail;
  blockstpb recover;

  trimparams() {
    ntrims         =       399;
    seed.blocks    =      1024;
    seed.tpb       =  SEED_TPB;
    trim0.blocks   =    NX2/NA;
    trim0.tpb      = TRIM0_TPB;
    trim1.blocks   =    NX2/NA;
    trim1.tpb      = TRIM1_TPB;
    trim.blocks    =       NX2;
    trim.tpb       =  TRIM_TPB;
    tail.blocks    =       NX2;
    tail.tpb       =  TAIL_TPB;;
    recover.blocks =      2048;
    recover.tpb    =       256;
  }
};

typedef u32 proof[PROOFSIZE];

// maintains set of trimmable edges
struct edgetrimmer {
  trimparams tp;
  edgetrimmer *dt;
  const size_t sizeA = ROW_EDGES_A * NX * sizeof(uint2);
  const size_t sizeB = ROW_EDGES_B * NX * sizeof(uint2);
  const size_t bufferSize = sizeB / NA + sizeA;
  const size_t indexesSize = NX2 * sizeof(u32);
  const size_t indexesSizeNA = NA * indexesSize;
  const size_t nodemapSize = NNODES / 8; // 8 bits per byte
  u8 *bufferA;
  u8 *bufferB;
  u8 *bufferA1;
  u32 *indexesA;
  u32 *indexesB;
  u32 *nodemap;
  u32 nedges;
  siphash_keys sipkeys;
  bool abort;
  bool initsuccess = false;

  edgetrimmer(const trimparams _tp) : tp(_tp) {
    checkCudaErrors_V(hipMalloc((void**)&dt, sizeof(edgetrimmer)));
    checkCudaErrors_V(hipMalloc((void**)&indexesA, indexesSizeNA));
    checkCudaErrors_V(hipMalloc((void**)&indexesB, indexesSizeNA));
    checkCudaErrors_V(hipMalloc((void**)&nodemap, nodemapSize));
    const size_t sizeC = ROW_EDGES_C * NX * sizeof(uint2);
    assert(bufferSize >= sizeB + sizeC);
    checkCudaErrors_V(hipMalloc((void**)&bufferB, bufferSize));
    bufferA = bufferB + sizeB / NA;
    bufferA1 = bufferB + sizeB;
    hipMemcpy(dt, this, sizeof(edgetrimmer), hipMemcpyHostToDevice);
    initsuccess = true;
  }
  u64 globalbytes() const {
    return bufferSize + 2 * indexesSizeNA + nodemapSize + sizeof(siphash_keys) + sizeof(edgetrimmer);
  }
  ~edgetrimmer() {
    checkCudaErrors_V(hipFree(bufferB));
    checkCudaErrors_V(hipFree(indexesA));
    checkCudaErrors_V(hipFree(indexesB));
    checkCudaErrors_V(hipFree(nodemap));
    checkCudaErrors_V(hipFree(dt));
    hipDeviceReset();
  }
  void indexcount(u32 round, const u32 *indexes) {
#ifdef VERBOSE
    u32 nedges[NX2];
    hipMemcpy(nedges, indexes, NX2 * sizeof(u32), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    u32 sum, max;
    for (int i = sum = max = 0; i < NX2; i++) {
      sum += nedges[i];
      if (nedges[i] > max)
        max = nedges[i];
    }
    print_log("round %d edges avg %d max %d\n", round, sum/NX2, max);
#endif
  }
  u32 trim() {
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start)); checkCudaErrors(hipEventCreate(&stop));
    hipMemcpyToSymbol(HIP_SYMBOL(dipkeys), &sipkeys, sizeof(sipkeys));

    hipDeviceSynchronize();
    float durationA, durationB, durationY;
    hipEventRecord(start, NULL);
  
    const u32 qI = NX2 / NA;
#if 1
    hipMemset(indexesA, 0, indexesSizeNA);
    for (u32 i=0; i < NA; i++) {
      YSeed<SEED_TPB, EDGES_A/NA><<<tp.seed.blocks, SEED_TPB>>>((uint2*)bufferA, indexesA+i*NX2, i);
      if (abort) return false;
    }
    hipDeviceSynchronize();
#ifdef VERBOSE
    print_log("%d x YSeed<<<%d,%d>>>\n", NA, tp.seed.blocks, tp.seed.tpb); // 1024x256
    indexcount(0, indexesA);
#endif
    hipMemset(nodemap, 0, nodemapSize);
    NodemapRound<TRIM0_TPB, EDGES_A/NA><<<NX2, TRIM0_TPB>>>((u32*)bufferA, indexesA, nodemap);
    if (abort) return false;

    checkCudaErrors(hipDeviceSynchronize()); hipEventRecord(stop, NULL);
    hipEventSynchronize(stop); hipEventElapsedTime(&durationY, start, stop);
    hipEventRecord(start, NULL);
  
#ifdef VERBOSE
    print_log("NodemapRound<<<%d,%d>>>\n", NX2/NA, TRIM0_TPB); // 1024x1024
    print_log("YSeeding completed in %.0f ms\n", durationY);
#endif

#endif

    hipMemset(indexesA, 0, indexesSizeNA);
    for (u32 i=0; i < NA; i++) {
      FluffySeed<SEED_TPB, EDGES_A/NA><<<tp.seed.blocks, SEED_TPB>>>((uint4*)(bufferA+i*(sizeA/NA2)), indexesA+i*NX2, i*(NEDGES/NA));
      if (abort) return false;
    }
  
#ifdef VERBOSE
    print_log("%d x Seed<<<%d,%d>>>\n", NA, tp.seed.blocks, tp.seed.tpb); // 1024x512
    indexcount(0, indexesA);
#endif

    checkCudaErrors(hipDeviceSynchronize()); hipEventRecord(stop, NULL);
    hipEventSynchronize(stop); hipEventElapsedTime(&durationA, start, stop);
    hipEventRecord(start, NULL);
  
#ifdef VERBOSE
    print_log("Seeding completed in %.0f ms\n", durationA);
    print_log("Round_A1<<<%d,%d>>>\n", NX2/NA, TRIM0_TPB); // 1024x1024
#endif

    hipMemset(indexesB, 0, indexesSizeNA);
    const u32 qB = sizeB/NA;
    for (u32 i=0; i < NA; i++) {
      FluffyRound_A1<TRIM0_TPB, EDGES_A/NA, EDGES_B/NA><<<NX2/NA, TRIM0_TPB>>>((uint2*)bufferA, (uint4*)(bufferB+i*qB), indexesA, indexesB, nodemap, i*qI); // .632
      if (abort) return false;
    }

    checkCudaErrors(hipDeviceSynchronize()); hipEventRecord(stop, NULL);
    hipEventSynchronize(stop); hipEventElapsedTime(&durationB, start, stop);
    checkCudaErrors(hipEventDestroy(start)); checkCudaErrors(hipEventDestroy(stop));
  
#ifdef VERBOSE
    indexcount(1, indexesB);
    print_log("Round A1 completed in %.0f ms\n", durationB);
    print_log("Round_A3<<<%d,%d>>>\n", NX2/NA, TRIM1_TPB); // 4096x1024
#endif

    hipMemset(indexesA, 0, indexesSize);
    FluffyRound_A3<TRIM1_TPB, EDGES_B/NA, EDGES_C><<<NX2, TRIM1_TPB>>>((uint2*)bufferB, (uint2*)bufferA1, indexesB, indexesA, nodemap);
    indexcount(2, indexesA); // .400
    if (abort) return false;

#ifdef VERBOSE
    print_log("Round_A2<><<<%d,%d>>>\n", NX2, TRIM_TPB); // 4096x512
#endif

    hipMemset(indexesB, 0, indexesSize);
    FluffyRound_A2<TRIM_TPB, EDGES_C, EDGES_B/2><<<NX2, TRIM_TPB>>>((uint2*)bufferA1, (uint2*)bufferB, indexesA, indexesB, nodemap);
    indexcount(3, indexesB);
    if (abort) return false;

    hipMemset(indexesA, 0, indexesSize);
    FluffyRound_A2<TRIM_TPB, EDGES_B/2, EDGES_A/4><<<NX2, TRIM_TPB>>>((uint2*)bufferB, (uint2*)bufferA1, indexesB, indexesA, nodemap);
    indexcount(4, indexesA);
    if (abort) return false;

    hipMemset(indexesB, 0, indexesSize);
    FluffyRound_A2<TRIM_TPB, EDGES_A/4, EDGES_A/4><<<NX2, TRIM_TPB>>>((uint2*)bufferA1, (uint2*)bufferB, indexesA, indexesB, nodemap);
    indexcount(5, indexesB);
    if (abort) return false;

    hipMemset(indexesA, 0, indexesSize);
    FluffyRound_A2<TRIM_TPB, EDGES_A/4, EDGES_B/4><<<NX2, TRIM_TPB>>>((uint2*)bufferB, (uint2*)bufferA1, indexesB, indexesA, nodemap);
    indexcount(6, indexesA);
    if (abort) return false;

    hipMemset(indexesB, 0, indexesSize);
    FluffyRound_A2<TRIM_TPB, EDGES_B/4, EDGES_B/4><<<NX2, TRIM_TPB>>>((uint2*)bufferA1, (uint2*)bufferB, indexesA, indexesB, nodemap);
    indexcount(7, indexesB);
    if (abort) return false;

    hipMemset(indexesA, 0, indexesSize);
    FluffyRound_A2<TRIM_TPB, EDGES_B/4, EDGES_A/8><<<NX2, TRIM_TPB>>>((uint2*)bufferB, (uint2*)bufferA1, indexesB, indexesA, nodemap);
    indexcount(8, indexesA);
    if (abort) return false;

    for (int round = 9; round < tp.ntrims; round += 2) {
      hipMemset(indexesB, 0, indexesSize);
      FluffyRound_A2<TRIM_TPB, EDGES_A/8, EDGES_A/8><<<NX2, TRIM_TPB>>>((uint2*)bufferA1, (uint2*)bufferB, indexesA, indexesB, nodemap);
      indexcount(round, indexesB);
      if (abort) return false;

      hipMemset(indexesA, 0, indexesSize);
      FluffyRound_A2<TRIM_TPB, EDGES_A/8, EDGES_A/8><<<NX2, TRIM_TPB>>>((uint2*)bufferB, (uint2*)bufferA1, indexesB, indexesA, nodemap);
      indexcount(round+1, indexesA);
      if (abort) return false;
    }
    
    hipMemset(indexesB, 0, indexesSize);
#ifdef VERBOSE
    print_log("Tail<><<<%d,%d>>>\n", NX2, TAIL_TPB);
#endif
    FluffyTail<TAIL_TPB, EDGES_A/8><<<NX2, TAIL_TPB>>>((uint2*)bufferA1, (uint2*)bufferB, indexesA, indexesB);

    hipMemcpy(&nedges, indexesB, sizeof(u32), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    print_log("%d rounds %d edges\n", tp.ntrims, nedges);
    return nedges;
  }
};

struct solver_ctx {
  edgetrimmer trimmer;
  bool mutatenonce;
  uint2 *edges;
  graph<word_t> cg;
  uint2 soledges[PROOFSIZE];
  std::vector<u32> sols; // concatenation of all proof's indices

  solver_ctx(const trimparams tp, bool mutate_nonce) : trimmer(tp), cg(MAXEDGES, MAXEDGES, MAX_SOLS, IDXSHIFT) {
    edges   = new uint2[MAXEDGES];
    mutatenonce = mutate_nonce;
  }

  void setheadernonce(char * const headernonce, const u32 len, const u32 nonce) {
    if (mutatenonce)
      ((u32 *)headernonce)[len/sizeof(u32)-1] = htole32(nonce); // place nonce at end
    setheader(headernonce, len, &trimmer.sipkeys);
    sols.clear();
  }
  ~solver_ctx() {
    delete[] edges;
  }

  int findcycles(uint2 *edges, u32 nedges) {
    cg.reset();
    for (u32 i = 0; i < nedges; i++) {
      cg.add_compress_edge(edges[i].x, edges[i].y);
    }
    for (u32 s = 0 ;s < cg.nsols; s++) {
#ifdef VERBOSE
      print_log("Solution");
#endif
      for (u32 j = 0; j < PROOFSIZE; j++) {
        soledges[j] = edges[cg.sols[s][j]];
#ifdef VERBOSE
        print_log(" (%x, %x)", soledges[j].x, soledges[j].y);
#endif
      }
#ifdef VERBOSE
      print_log("\n");
#endif
      sols.resize(sols.size() + PROOFSIZE);
      hipMemcpyToSymbol(HIP_SYMBOL(recovery), soledges, sizeof(soledges));
      hipMemset(trimmer.indexesA, 0, trimmer.indexesSize);
#ifdef VERBOSE
    print_log("Recovery<><<<%d,%d>>>\n", trimmer.tp.recover.blocks, trimmer.tp.recover.tpb);
#endif
      FluffyRecovery<<<trimmer.tp.recover.blocks, trimmer.tp.recover.tpb>>>((u32 *)trimmer.bufferA1);
      hipMemcpy(&sols[sols.size()-PROOFSIZE], trimmer.bufferA1, PROOFSIZE * sizeof(u32), hipMemcpyDeviceToHost);
      checkCudaErrors(hipDeviceSynchronize());
      qsort(&sols[sols.size()-PROOFSIZE], PROOFSIZE, sizeof(u32), cg.nonce_cmp);
    }
    return 0;
  }

  int solve() {
    u64 time0, time1;
    u32 timems,timems2;

    trimmer.abort = false;
    time0 = timestamp();
    u32 nedges = trimmer.trim();
    if (!nedges)
      return 0;
    if (nedges > MAXEDGES) {
      print_log("OOPS; losing %d edges beyond MAXEDGES=%d\n", nedges-MAXEDGES, MAXEDGES);
      nedges = MAXEDGES;
    }
    hipMemcpy(edges, trimmer.bufferB, sizeof(uint2[nedges]), hipMemcpyDeviceToHost);
    time1 = timestamp(); timems  = (time1 - time0) / 1000000;
    time0 = timestamp();
    findcycles(edges, nedges);
    time1 = timestamp(); timems2 = (time1 - time0) / 1000000;
    print_log("trim time %d ms findcycles edges %d time %d ms total %d ms\n", timems, nedges, timems2, timems+timems2);
    return sols.size() / PROOFSIZE;
  }

  void abort() {
    trimmer.abort = true;
  }
};

#include <unistd.h>

// arbitrary length of header hashed into siphash key
#define HEADERLEN 80

typedef solver_ctx SolverCtx;

CALL_CONVENTION int run_solver(SolverCtx* ctx,
                               char* header,
                               int header_length,
                               u32 nonce,
                               u32 range,
                               SolverSolutions *solutions,
                               SolverStats *stats
                               )
{
  u64 time0, time1;
  u32 timems;
  u32 sumnsols = 0;
  int device_id;
  if (stats != NULL) {
    hipGetDevice(&device_id);
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, stats->device_id);
    stats->device_id = device_id;
    stats->edge_bits = EDGEBITS;
    strncpy(stats->device_name, props.name, MAX_NAME_LEN);
  }

  if (ctx == NULL || !ctx->trimmer.initsuccess){
    print_log("Error initialising trimmer. Aborting.\n");
    print_log("Reason: %s\n", LAST_ERROR_REASON);
    if (stats != NULL) {
       stats->has_errored = true;
       strncpy(stats->error_reason, LAST_ERROR_REASON, MAX_NAME_LEN);
    }
    return 0;
  }

  for (u32 r = 0; r < range; r++) {
    time0 = timestamp();
    ctx->setheadernonce(header, header_length, nonce + r);
    print_log("nonce %d k0 k1 k2 k3 %llx %llx %llx %llx\n", nonce+r, ctx->trimmer.sipkeys.k0, ctx->trimmer.sipkeys.k1, ctx->trimmer.sipkeys.k2, ctx->trimmer.sipkeys.k3);
    u32 nsols = ctx->solve();
    time1 = timestamp();
    timems = (time1 - time0) / 1000000;
    print_log("Time: %d ms\n", timems);
    for (unsigned s = 0; s < nsols; s++) {
      print_log("Solution");
      u32* prf = &ctx->sols[s * PROOFSIZE];
      for (u32 i = 0; i < PROOFSIZE; i++)
        print_log(" %jx", (uintmax_t)prf[i]);
      print_log("\n");
      if (solutions != NULL){
        solutions->edge_bits = EDGEBITS;
        solutions->num_sols++;
        solutions->sols[sumnsols+s].nonce = nonce + r;
        for (u32 i = 0; i < PROOFSIZE; i++) 
          solutions->sols[sumnsols+s].proof[i] = (u64) prf[i];
      }
      int pow_rc = verify(prf, ctx->trimmer.sipkeys);
      if (pow_rc == POW_OK) {
        print_log("Verified with cyclehash ");
        unsigned char cyclehash[32];
        blake2b((void *)cyclehash, sizeof(cyclehash), (const void *)prf, sizeof(proof), 0, 0);
        for (int i=0; i<32; i++)
          print_log("%02x", cyclehash[i]);
        print_log("\n");
      } else {
        print_log("FAILED due to %s\n", errstr[pow_rc]);
      }
    }
    sumnsols += nsols;
    if (stats != NULL) {
      stats->last_start_time = time0;
      stats->last_end_time = time1;
      stats->last_solution_time = time1 - time0;
    }
  }
  print_log("%d total solutions\n", sumnsols);
  return sumnsols > 0;
}

CALL_CONVENTION SolverCtx* create_solver_ctx(SolverParams* params) {
  trimparams tp;
  tp.ntrims = params->ntrims;
  tp.seed.blocks = params->genablocks;
  tp.seed.tpb = params->genatpb;
  tp.trim0.tpb = params->genbtpb;
  tp.trim.tpb = params->trimtpb;
  tp.tail.tpb = params->tailtpb;
  tp.recover.blocks = params->recoverblocks;
  tp.recover.tpb = params->recovertpb;

  hipDeviceProp_t prop;
  checkCudaErrors_N(hipGetDeviceProperties(&prop, params->device));

  assert(tp.seed.tpb <= prop.maxThreadsPerBlock);
  assert(tp.trim0.tpb <= prop.maxThreadsPerBlock);
  assert(tp.trim.tpb <= prop.maxThreadsPerBlock);
  // assert(tp.tailblocks <= prop.threadDims[0]);
  assert(tp.tail.tpb <= prop.maxThreadsPerBlock);
  assert(tp.recover.tpb <= prop.maxThreadsPerBlock);

  hipSetDevice(params->device);
  if (!params->cpuload)
    checkCudaErrors_N(hipSetDeviceFlags(hipDeviceScheduleBlockingSync));

  return new SolverCtx(tp, params->mutate_nonce);
}

CALL_CONVENTION void destroy_solver_ctx(SolverCtx* ctx) {
  delete ctx;
}

CALL_CONVENTION void stop_solver(SolverCtx* ctx) {
  ctx->abort();
}

CALL_CONVENTION void fill_default_params(SolverParams* params) {
  trimparams tp;
  params->device = 0;
  params->ntrims = tp.ntrims;
  params->genablocks = tp.seed.blocks;
  params->genatpb = tp.seed.tpb;
  params->genbtpb = tp.trim0.tpb;
  params->trimtpb = tp.trim.tpb;
  params->tailtpb = tp.tail.tpb;
  params->recoverblocks = tp.recover.blocks;
  params->recovertpb = tp.recover.tpb;
  params->cpuload = false;
}


static_assert(NLISTS % (RELAY_TPB) == 0);    // for Tag_Edges lists    init
static_assert(NZ % (32 * TRIM0_TPB) == 0); // for Round_A1 ecounters init
static_assert(NZ % (32 * TRIM1_TPB) == 0); // for Round_A3 ecounters init
static_assert(NZ % (32 *  TRIM_TPB) == 0); // for Round_A2 ecounters init

int main(int argc, char **argv) {
  trimparams tp;
  u32 nonce = 0;
  u32 range = 1;
  u32 device = 0;
  char header[HEADERLEN];
  u32 len;
  int c;

  // set defaults
  SolverParams params;
  fill_default_params(&params);

  memset(header, 0, sizeof(header));
  while ((c = getopt(argc, argv, "scd:h:m:n:r:U:Z:z:")) != -1) {
    switch (c) {
      case 's':
        print_log("SYNOPSIS\n  cuda%d [-s] [-c] [-d device] [-h hexheader] [-m trims] [-n nonce] [-r range] [-U seedblocks] [-Z recoverblocks] [-z recoverthreads]\n", EDGEBITS);
        print_log("DEFAULTS\n  cuda%d -d %d -h \"\" -m %d -n %d -r %d -U %d -Z %d -z %d\n", EDGEBITS, device, tp.ntrims, nonce, range, tp.seed.blocks, tp.recover.blocks, tp.recover.tpb);
        exit(0);
      case 'c':
        params.cpuload = false;
        break;
      case 'd':
        device = params.device = atoi(optarg);
        break;
      case 'h':
        len = strlen(optarg)/2;
        assert(len <= sizeof(header));
        for (u32 i=0; i<len; i++)
          sscanf(optarg+2*i, "%2hhx", header+i); // hh specifies storage of a single byte
        break;
      case 'm': // ntrims         =       458;
        params.ntrims = atoi(optarg);
        break;
      case 'n':
        nonce = atoi(optarg);
        break;
      case 'r':
        range = atoi(optarg);
        break;
      case 'U': // seed.blocks    =      1024;
        params.genablocks = atoi(optarg);
        break;
      case 'Z': // recover.blocks =      2048;
        params.recoverblocks = atoi(optarg);
        break;
      case 'z': // recover.tpb    =       256;
        params.recovertpb = atoi(optarg);
        break;
    }
  }

  int nDevices;
  checkCudaErrors(hipGetDeviceCount(&nDevices));
  assert(device < nDevices);
  hipDeviceProp_t prop;
  checkCudaErrors(hipGetDeviceProperties(&prop, device));
  u64 dbytes = prop.totalGlobalMem;
  int dunit;
  for (dunit=0; dbytes >= 102040; dbytes>>=10,dunit++) ;
  print_log("%s with %d%cB @ %d bits x %dMHz\n", prop.name, (u32)dbytes, " KMGT"[dunit], prop.memoryBusWidth, prop.memoryClockRate/1000);
  // hipSetDevice(device);

  print_log("Looking for %d-cycle on cuckaroom%d(\"%s\",%d", PROOFSIZE, EDGEBITS, header, nonce);
  if (range > 1)
    print_log("-%d", nonce+range-1);
  print_log(") with 50%% edges, %d*%d buckets, %d trims, and %d thread blocks.\n", NX, NY, params.ntrims, NX);

  assert(params.recovertpb >= PROOFSIZE);
  SolverCtx* ctx = create_solver_ctx(&params);

  u64 bytes = ctx->trimmer.globalbytes();
  int unit;
  for (unit=0; bytes >= 102400; bytes>>=10,unit++) ;
  print_log("Using %d%cB of global memory.\n", (u32)bytes, " KMGT"[unit]);

  run_solver(ctx, header, sizeof(header), nonce, range, NULL, NULL);

  return 0;
}
